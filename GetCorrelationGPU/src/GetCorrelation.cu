#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<fitsio.h>
#include<pil.h>
#include<GetCorrelation.h>
#include"svdfit.c"
/*
 *  nvcc GetCorrelation.cu  $INCL $OTHERLIB -arch=sm_20
 *
 */
int main(int argc,char*argv[])
{
	//declaring the local variables
	char parfilename[BUFSIZE],dphfilename[BUFSIZE],correlationFilename[BUFSIZE],qeFilename[BUFSIZE],badpixfilename[BUFSIZE];
	int moddph_size=sizeof(float)*NUMPIXPERMODULE*NUMPIXPERMODULE;
	float *mod_dph,*qemap,*dphWithoutBadPix;
	int moduleid,llx,lly,numelements_x,numelements_y;
	int i=0,j=0,rv=0,ii=0,jj=0,numbadpix=0;
	int *mask,*badpix,*quad_dph,pixid=0;
	float x_center,y_center,*correlation,zs;// xs,ys*shadow,
	float xstart=-19.5,ystart=-19.5,xend=19.5,yend=19.5;
	float resol=1,mean,rms;
	time_t start;
	int tempsrc=0;
	float *dph_device,*correlation_device=NULL,*qemap_device=NULL;
	float fittedparam=0;
	int *mask_device=NULL,numblocks-0,threadsPerBlock=0;
	FILE *fp;
	start=clock();
	char  *CZTHOME;
	hipError_t cuerr;
	float maxvalue=0;
	int max_x=0,max_y=0;
	void (*funx_pointer)(float,float*)=polyfunction;
	float *funy,*funxy,*coeff,chisq,*shadow;
	float xs,ys,minval=9999999999,min_x=0,min_y=0,xt_tmp=xstart,yt_tmp=ystart,xe_tmp=xend,ye_tmp=yend;
	int fitcounter=0;


	//getting path for CZTI environment
	CZTHOME = getenv ("CZTWORKSPACE");
	if(CZTHOME==NULL)
	{
		printf("CZTHOME Variable is not set\n");
		exit(0);
	}
	//Setting par file name
	strcpy(parfilename,CZTHOME);
	strcat(parfilename,"/GetCorrelationGPU/par/GetCorrelation");
	PILSetModuleName(parfilename);
	int r=PILInit(argc,argv);
	if(r<0)
	{
		printf("Error(%s:%d) : Error while loading par file\n",__FILE__,__LINE__);
		exit(0);
	}

	//reading inputs
	r=PILGetInt("moduleid",&moduleid);
	r=PILGetReal4("Zs",&zs);
	r=PILGetReal4("xstart",&xstart);
	r=PILGetReal4("xend",&xend);
	r=PILGetReal4("ystart",&ystart);
	r=PILGetReal4("yend",&yend);
	r=PILGetReal4("resolution",&resol);
	r=PILGetFname("dphfilename",dphfilename);
	r=PILGetFname("badpixfilename",badpixfilename);
	r=PILGetFname("qeFilename",qeFilename);
	r=PILGetFname("correlationfilename",correlationFilename);
	PILClose(r);

	//allocating the memory
	numelements_x=(int)(((xend-xstart)/resol)+1);
	numelements_y=(int)(((yend-ystart)/resol)+1);
	badpix=(int*)malloc(sizeof(int)*ROWSPERMODULE*COLSPERMODULE);
	mask=(int*)malloc(sizeof(int)*ROWSPERMODULE*COLSPERMODULE);
	correlation=(float*)malloc(sizeof(float)*numelements_y*numelements_x);
	quad_dph=(int*)malloc(sizeof(int)*NUMPIXPERQUAD*NUMPIXPERQUAD);
	mod_dph=(float*)malloc(moddph_size);
	qemap=(float*)malloc(sizeof(float)*NUMPIXPERMODULE*NUMPIXPERMODULE);
	dphWithoutBadPix=(float*)malloc(sizeof(float)*NUMPIXPERMODULE*NUMPIXPERMODULE);

	if(mask==NULL || badpix==NULL ||correlation==NULL ||quad_dph==NULL ||mod_dph==NULL ||qemap==NULL ||dphWithoutBadPix==NULL)
	{
		printf("Error while allocating memory\n");
		exit(0);
	}
	//Reading DPH, QE files
	readImage(dphfilename,quad_dph,2);
	readFloatImage(qeFilename,2,qemap);
	getminmax(moduleid,&llx,&lly,&x_center,&y_center);
	getModule(moduleid+1,mask);
	
	//initilizing the arrays
	for(i=0;i<numelements_x;i++)
	{
		for(j=0;j<numelements_y;j++)
			correlation[j*NUMPIXPERMODULE+i]=0;
	}
	for(i=llx,ii=0;i<llx+NUMPIXPERMODULE;i++,ii++)
	{
		for(j=lly,jj=0;j<lly+NUMPIXPERMODULE;j++,jj++)
			mod_dph[jj*NUMPIXPERMODULE+ii]=quad_dph[j*NUMPIXPERQUAD+i];
	}
	
	//reading bad pixel list
	fp=fopen(badpixfilename,"r");
	if(fp==NULL)
	{
		printf("%s file not found\n",badpixfilename);
		exit(0);
	}
	while(1)
	{
		rv=fscanf(fp,"%d",&badpix[numbadpix]);
		if(rv==-1)
			break;
		numbadpix++;
	}
	//here assign -1 for the bad pixels so they can be ignored from computations
	int isbad=0;
	for(i=0;i<NUMPIXPERMODULE;i++)
	{
		for(j=0,isbad=0;j<NUMPIXPERMODULE;j++)
		{
			pixid=(15-i)*16+j;
			for(ii=0;ii<numbadpix;ii++)
			{
					if(pixid==badpix[ii])
					{
						isbad=1;break;
					}
			}
			if(isbad)
			{
				dphWithoutBadPix[pixid]=-1;
			}
			else
			{
				dphWithoutBadPix[pixid]=mod_dph[pixid];
			}
		}
	}

	
	numblocks=(int)ceil(((numelements_x*numelements_y)/(float)NUMTHREADS));
	printf("NUM BLOCKS:%d\t",numblocks);
	printf("Threads per block:%d\n",threadsPerBlock);

	//allocating device memory
	hipMalloc((void **) &dph_device,moddph_size );
	hipMalloc((void **) &qemap_device,moddph_size );
	hipMalloc((void **) &mask_device,sizeof(int)*ROWSPERMODULE*COLSPERMODULE );
	hipMalloc((void **) &correlation_device, sizeof(float)*numelements_x*numelements_y);
	//checking error in memory allocation
	if((cuerr = hipGetLastError()) != hipSuccess)
	{
		printf("\nError: Cuda Malloc %s\n", hipGetErrorString(cuerr));
		return -1;
	}
	//coping data from host to device memory
	hipMemcpy(dph_device,dphWithoutBadPix,moddph_size,hipMemcpyHostToDevice);
	hipMemcpy(qemap_device,qemap,moddph_size,hipMemcpyHostToDevice);
	hipMemcpy(mask_device,mask,sizeof(int)*ROWSPERMODULE*COLSPERMODULE ,hipMemcpyHostToDevice);
	if((cuerr = hipGetLastError()) != hipSuccess)
	{
		printf("\nError: Cuda Memcpy %s\n", hipGetErrorString(cuerr));
		return -1;
	}
	
	//calling cross-correlation kernel
	CorrelationKernel<<<numblocks,threadsPerBlock>>>(mask_device,dph_device,qemap_device,correlation_device,numelements_x,numelements_y,xstart,ystart,resol,zs,moduleid);
	if((cuerr = hipGetLastError()) != hipSuccess)
	{
		printf("\nError: CUDA KERNEL ERROR: %s\n", hipGetErrorString(cuerr));
		return -1;
	}
	//coping back the results from device to host memory
	hipMemcpy(correlation,correlation_device,sizeof(float)*numelements_x*numelements_y,hipMemcpyDeviceToHost);
	if((cuerr = hipGetLastError()) != hipSuccess)
	{
		printf("\nError: Cuda DEVICE TO HOST Memcpy: %s\n", hipGetErrorString(cuerr));
		return -1;
	}
	//getting maximum cross-correlation value and preparing array to write FITS image 
	for(i=0;i<numelements_x;i++)
	{	for(j=0;j<numelements_y;j++)
		{
			if(correlation[i*numelements_y+j]>maxvalue){
				maxvalue=correlation[i*numelements_y+j];
				max_x=i;
				max_y=j;
			}
		}
	}
	printf("Max_x:%d\tMax_y:%d\n",max_x,max_y);
	printf("Peak:%f\tX:%f\tY:%f\n",maxvalue,xstart+(max_x*resol),ystart+(max_y*resol));
	writeFloatImage(correlationFilename,correlation,numelements_x,numelements_y,resol);
	
	//performing forward fitting to get source intensity
	shadow=(float*)malloc(sizeof(float)*(NUMPIXPERMODULE*NUMPIXPERMODULE)+2);
	funy=(float*)malloc(sizeof(float)*(NUMPIXPERMODULE*NUMPIXPERMODULE)+2);
	funxy=(float*)malloc(sizeof(float)*(NUMPIXPERMODULE*NUMPIXPERMODULE)+2);
	coeff=(float*)malloc(sizeof(float)*(NUMPIXPERMODULE*NUMPIXPERMODULE)+2);
	xstart=xstart+(max_x*resol);
	ystart=ystart+(max_y*resol);
	int windowsize=3;
	printf("Fitting source for iteration :%d\n",tempsrc);
	for(xs=xstart-windowsize;xs<=xstart+windowsize;xs+=0.25)
	{
		for(ys=ystart-windowsize;ys<=ystart+windowsize;ys+=0.25)
		{
			gensha(moduleid,shadow,xs,ys,zs,mask);
			fitcounter=0;
			for(i=0;i<NUMPIXPERMODULE*NUMPIXPERMODULE;i++){
				if(dphWithoutBadPix[i]!=-1)
				{
					funy[fitcounter+1]=shadow[i];
					funxy[fitcounter+1]=dphWithoutBadPix[i];
					fitcounter++;
				}
			}
			do_svdfit(funy,funxy,coeff,fitcounter,1,&chisq,funx_pointer);
			chisq/=fitcounter-1;
			if(chisq<minval)
			{
				minval=chisq;
				fittedparam=coeff[1];
				min_x=xs;
				min_y=ys;
			}
		}
	}
	char splot[BUFSIZ],matrix[BUFSIZE],xplot[BUFSIZE],yplot[BUFSIZE];
	FILE *fp1,*fp2,*fp3,*fp4;

	strcpy(splot,correlationFilename);
	strcpy(matrix,correlationFilename);
	strcpy(xplot,correlationFilename);
	strcpy(yplot,correlationFilename);

	strcat(splot,"_S.txt");
	strcat(matrix,"_Corr.txt");
	strcat(xplot,"_X.txt");
	strcat(yplot,"_Y.txt");

	fp1=fopen(splot,"w");
	fp2=fopen(matrix,"w");
	fp3=fopen(xplot,"w");
	fp4=fopen(yplot,"w");
	for(xs=xt_tmp,i=0;xs<=xe_tmp;xs+=0.25,i++)
	{
		for(ys=yt_tmp,j=0;ys<=ye_tmp;ys+=0.25,j++)
		{
			fprintf(fp1,"%f\t%f\t%f\n",xs,ys,correlation[i*numelements_y+j]);
			fprintf(fp2,"%f\t",correlation[i*numelements_y+j]);
			fprintf(fp3,"%f\t%f\n",xs,correlation[i*numelements_y+j]);
			fprintf(fp4,"%f\t%f\n",ys,correlation[i*numelements_y+j]);
		}
		fprintf(fp2,"\n");
	}
	fclose(fp1);
	fclose(fp2);
	fclose(fp3);
	fclose(fp4);

	printf("Minimum Chisq is %f found at %f,%f\n",minval,min_x,min_y);
	printf("Fitted Param:%f\n",fittedparam);
	for(i=0;i<NUMPIXPERMODULE*NUMPIXPERMODULE;i++){
		if(dphWithoutBadPix[i]!=-1)
		{
			dphWithoutBadPix[i]-=shadow[i]*fittedparam;
		}
	}



	printf("TIME ELAPSED : %f seconds\n\n", ((double)clock() - start) / CLOCKS_PER_SEC);
}
void polyfunction(float x,float*funx)
{
	funx[1]=x;
}
__global__ void CorrelationKernel(int *mask,float *dph,float *qemap,float *correlation,int numelem_x,int numelem_y,float x_start,float y_start,float resol,float zs,int moduleid)
{
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	int i=0;
	float xs,ys;
	float *shadow,sum=0;
	xs=(x_start)+((index/numelem_y)*resol);
	ys=(y_start)+((index%numelem_y)*resol);
	if(xs>19.5 || xs<-19.5 || ys>19.5 || ys<-19.5)
		return;
	//allocating memory
	shadow=(float*)malloc(sizeof(float)*ELEM);
	//computing shadow
	gensha_device(moduleid,shadow,xs,ys,zs,mask);
	
	//Computing cross-correlation value for a direction
	for(i=0;i<NUMPIXPERMODULE*NUMPIXPERMODULE;i++)
	{
		if(qemap[i]!=0 && dph[i]!=-1)
			sum+=((dph[i]*shadow[i])/qemap[i]);
	}
	correlation[index]=sum;
	free(shadow);
}

//function to generate shadow
int gensha(int moduleid,float*det,float xs,float ys,float zs,int *mask)
{
	int detx=0, dety=0;
	float x_center=0.0,y_center=0.0,tx=0.0,ty=0.0,xtemp=0,ytemp=0;
	int rowcounter=0,colcounter=0, x_index,y_index;
	int i=0,j=0,ii=0,jj=0;
	int llx=0,lly=0;


	getminmax(moduleid,&llx,&lly,&x_center,&y_center);
	dety=(llx/16)*ROWSPERMODULE+(llx/16)*100;
	detx=(lly/16)*COLSPERMODULE+(lly/16)*100;

	xtemp=19.5-xs;
	ytemp=19.5+ys;
	xs=ytemp;
	ys=xtemp;
	xs/=0.02;
	ys/=0.02;
	zs/=0.02;
	int tempx,tempy;
	for(i=0;i<256;i++)
			det[i]=0;
	for(i=detx,ii=0;i<detx+ROWSPERMODULE;i++,ii++)
	{
		tx=asin( (ys-ii) / (  sqrt( (zs*zs)+((ys-ii) * (ys-ii)) )  ) );
		tempx=(int)(ii+(HEIGHT*(tan(tx))));
		x_index=(ii/123);
		if(x_index>14)
			x_index=14+((ii-(14*123))/114);
		else
			x_index=x_index+((ii-(x_index*123))/114);
		for(j=dety,jj=0;j<dety+COLSPERMODULE;j++,jj++)
		{
			y_index=(jj/123);
			if(y_index>14)
				y_index=14+((jj-(14*123))/114);
			else
				y_index=y_index+((jj-(y_index*123))/114);
			ty=asin( (xs-jj) / (  sqrt( (zs*zs)+((xs-jj) * (xs-jj)) )  ) );
			tempy=(int)(jj+(HEIGHT*(tan(ty))));
			if(tempx>=0 && tempx<ROWSPERMODULE && tempy>=0 &&tempy<COLSPERMODULE)
			{
				if(mask[tempx*COLSPERMODULE+tempy]==1)
					det[x_index*16+y_index]+=1;
			}
		}
	}
	for(i=0;i<16;i++)
	{
		if(i%16==0 ||(i+1)%16==0)
			rowcounter=114;
		else
			rowcounter=123;
		for(j=0;j<16;j++)
		{
			if(j%16==0 || (j+1)%16==0)
				colcounter=114;
			else
				colcounter=123;
			det[i*16+j]=(float)det[i*16+j]/(float)(rowcounter*colcounter);
		}
	}

	return 0;
}

//Generatee shadow function for devicee
__device__ int gensha_device(int moduleid,float*det,float xs,float ys,float zs,int *mask)
{
	int detx=0, dety=0;
	float x_center=0.0,y_center=0.0,tx=0.0,ty=0.0,xtemp=0,ytemp=0;
	int rowcounter=0,colcounter=0, x_index,y_index;
	int i=0,j=0,ii=0,jj=0;
	int llx=0,lly=0;


	getminmax_device(moduleid,&llx,&lly,&x_center,&y_center);
	dety=(llx/16)*ROWSPERMODULE+(llx/16)*100;
	detx=(lly/16)*COLSPERMODULE+(lly/16)*100;

	xtemp=19.5-xs;
	ytemp=19.5+ys;
	xs=ytemp;
	ys=xtemp;
	xs/=0.02;
	ys/=0.02;
	zs/=0.02;
	int tempx,tempy;
	for(i=0;i<256;i++)
			det[i]=0;
	for(i=detx,ii=0;i<detx+ROWSPERMODULE;i++,ii++)
	{
		tx=asin( (ys-ii) / (  sqrt( (zs*zs)+((ys-ii) * (ys-ii)) )  ) );
		tempx=(int)(ii+(HEIGHT*(tan(tx))));
		x_index=(ii/123);
		if(x_index>14)
			x_index=14+((ii-(14*123))/114);
		else
			x_index=x_index+((ii-(x_index*123))/114);
		for(j=dety,jj=0;j<dety+COLSPERMODULE;j++,jj++)
		{
			y_index=(jj/123);
			if(y_index>14)
				y_index=14+((jj-(14*123))/114);
			else
				y_index=y_index+((jj-(y_index*123))/114);
			ty=asin( (xs-jj) / (  sqrt( (zs*zs)+((xs-jj) * (xs-jj)) )  ) );
			tempy=(int)(jj+(HEIGHT*(tan(ty))));
			if(tempx>=0 && tempx<ROWSPERMODULE && tempy>=0 &&tempy<COLSPERMODULE)
			{
				if(mask[tempx*COLSPERMODULE+tempy]==1)
					det[x_index*16+y_index]+=1;
			}
		}
	}
	for(i=0;i<16;i++)
	{
		if(i%16==0 ||(i+1)%16==0)
			rowcounter=114;
		else
			rowcounter=123;
		for(j=0;j<16;j++)
		{
			if(j%16==0 || (j+1)%16==0)
				colcounter=114;
			else
				colcounter=123;
			det[i*16+j]=(float)det[i*16+j]/(float)(rowcounter*colcounter);
		}
	}

	return 0;
}
//function to write fits images
void writeFloatImage(char *filename,float*pixels,int rows,int cols,float resol)
{
	int bitpix   =  FLOAT_IMG; /* 16-bit unsigned short pixel values       */
    long naxis    =   2;  /* 2-dimensional image                            */
    int fpixel = 1,status=0;
	long naxes[2] = { cols,rows };
	long nelements = naxes[0] * naxes[1];
	fitsfile *fptr;
	remove(filename);
	if (fits_create_file(&fptr, filename, &status))
	{
		printf("Error(%s:%d):Creating file\n",__FILE__,__LINE__);
	}

	if ( fits_create_img(fptr,  bitpix,0, naxes, &status) )
	{
		printf("Error(%s:%d):Creating image\n",__FILE__,__LINE__);
	}


	if ( fits_create_img(fptr,  bitpix, naxis, naxes, &status) )
	{
		printf("Error(%s:%d):Creating image\n",__FILE__,__LINE__);
	}
	write_wcsaxis(fptr,1,"","","","IMX",((double)(rows+1)/2.0),resol,0,"mm",&status);
	write_wcsaxis(fptr,2,"","","","IMY",((double)(cols+1)/2.0),resol,0,"mm",&status);
	if ( fits_write_img(fptr, TFLOAT, fpixel, nelements,pixels, &status) )
	{
			printf("Error(%s:%d):Wrting image\n",__FILE__,__LINE__);
	}
	if ( fits_close_file(fptr, &status) )
	{
			printf("Error(%s:%d):closing file\n",__FILE__,__LINE__);
	}
}

//function to read fits images
void readFloatImage(char*filename,int hduno,float*data)
{
		fitsfile *fptr;       /* pointer to the FITS file, defined in fitsio.h */
	    int status,  nfound, anynull,hdutype;
	    long naxes[2], fpixel, nbuffer, npixels;
	    float  nullval;
	    int buffsize;
	    status = 0;
	    if ( fits_open_file(&fptr, filename, READONLY, &status) )
	    	printf("Error while reading fits file\n");

	   if ( fits_movabs_hdu(fptr, hduno, &hdutype, &status) )
	           printf("Error while moving HDU\n");
	    if ( fits_read_keys_lng(fptr, "NAXIS", 1, 2, naxes, &nfound, &status) )
	    	  printf("Error while reading keys\n");
	    npixels  = naxes[0] * naxes[1];         /* number of pixels in the image */
	    buffsize=npixels;
	    fpixel   = 1;
	    nullval  = 0;                /* don't check for null values in the image */
	    while (npixels > 0)
	    {
	      nbuffer = npixels;
	      if (npixels > buffsize)
	        nbuffer = buffsize;
	      if ( fits_read_img(fptr, TFLOAT, fpixel, nbuffer, &nullval,data, &anynull, &status) )
	    	  printf("Error while reading fits image\n");
	      npixels -= nbuffer;
	      fpixel  += nbuffer;
	    }
	    if ( fits_close_file(fptr, &status) )
	    	  printf("Error while closing file\n");

}

//Writing WCS information to images
int write_wcsaxis(fitsfile *imgfile, int axis, char *suffix,char *wcsname, char *wcstype, char *ctype, double crpix, double cdelt, double crval,char *cunit, int *status)
{
	char key[20];

	if (status == 0) return NULL_INPUT_PTR;
	if (*status != 0) return (*status);
	if (imgfile == 0) return (*status = NULL_INPUT_PTR);

	if (wcsname && wcsname[0]) {
		sprintf(key, "WCSNAME%s", suffix);
		fits_update_key(imgfile, TSTRING, key, wcsname,"Coordinate system name", status);
	}
	if (wcstype && wcstype[0]){
		sprintf(key, "WCSTY%d%s", axis, suffix);
		fits_update_key(imgfile, TSTRING, key, wcstype,"Coordinate system axis", status);
	}
	sprintf(key, "CTYPE%d%s", axis, suffix);
	fits_update_key(imgfile, TSTRING, key, ctype,"Name of coordinate", status);

	if (cunit && cunit[0]) {
		sprintf(key, "CUNIT%d%s", axis, suffix);
		fits_update_key(imgfile, TSTRING, key, cunit,"Units of coordinate axis", status);

	}
	sprintf(key, "CRPIX%d%s", axis, suffix);
	fits_update_key(imgfile, TDOUBLE, key, &crpix,"Reference pixel position", status);
	sprintf(key, "CDELT%d%s", axis, suffix);
	fits_update_key(imgfile, TDOUBLE, key, &cdelt,"Pixel spacing in physical units", status);
	sprintf(key, "CRVAL%d%s", axis, suffix);
	fits_update_key(imgfile, TDOUBLE, key, &crval,"Coordinate value at reference pixel position", status);
	return (*status);
}

void normalizeData(float *data,int numelements)
{
	float peakval=getMaximum(data,numelements);
	int i=0;
	for(i=0;i<numelements;i++)
	{
		data[i]/=peakval;
	}
}
float getMaximum(float *data,int numelements)
{
	int i=0;
	float maxval=0;
	for(i=0;i<numelements;i++)
	{
		if(data[i]>maxval)
			maxval=data[i];
	}
	return maxval;
}

void readImage(char*filename,int *buffer,int hduNo)
{
	fitsfile *fptr;       /* pointer to the FITS file, defined in fitsio.h */
	int status,  nfound, anynull,hdutype;
	long naxes[2], fpixel, nbuffer, npixels;
	float nullval;
	status = 0;

	if ( fits_open_file(&fptr, filename, READONLY, &status) )
		printf("Error while opening fits file\n");

	if ( fits_movabs_hdu(fptr, hduNo, &hdutype, &status) )
		printf("Error while moving module\n");

	if ( fits_read_keys_lng(fptr, "NAXIS", 1, 2, naxes, &nfound, &status) )
		printf("Error while reading keys\n");

	npixels  = naxes[0] * naxes[1];         /* number of pixels in the image */
	fpixel   = 1;
	nullval  = 0;                /* don't check for null values in the image */
	while (npixels > 0)
	{
		nbuffer = npixels;
		if ( fits_read_img(fptr, TINT, fpixel, nbuffer, &nullval,buffer, &anynull, &status) )
		{
			printf("Error reading fits image\n");
		}
		npixels -= nbuffer;
		fpixel  += nbuffer;
	}
	if ( fits_close_file(fptr, &status) )
	{
		printf("Error while closing fits file\n");
	}
	return;
}
__device__ void getminmax_device(int moduleno,int*x,int *y,float* x_center,float* y_center)
{
	switch(moduleno)
	{
	case 12 :
		*x=0;
		*y=0;
		*x_center=19.5;
		*y_center=19.5;
		break;
	case 13 :
		*x=16;
		*y=0;
		*x_center=60.5;
		*y_center=19.5;
		break;
	case 14 :
		*x=32;
		*y=0;
		*x_center=101.5;
		*y_center=19.5;
		break;
	case 15 :
		*x=48;
		*y=0;
		*x_center=142.5;
		*y_center=19.5;
		break;
	case 8:
		*x=0;
		*y=16;
		*x_center=19.5;
		*y_center=60.5;
		break;
	case 9 :
		*x=16;
		*y=16;
		*x_center=60.5;
		*y_center=60.5;
		break;
	case 10 :
		*x=32;
		*y=16;
		*x_center=101.5;
		*y_center=60.5;
		break;
	case 11 :
		*x=48;
		*y=16;
		*x_center=142.5;
		*y_center=60.5;
		break;
	case 4 :
		*x=0;
		*y=32;
		*x_center=19.5;
		*y_center=101.5;
		break;
	case 5 :
		*x=16;
		*y=32;
		*x_center=60.5;
		*y_center=101.5;
		break;
	case 6 :
		*x=32;
		*y=32;
		*x_center=101.5;
		*y_center=101.5;
		break;
	case 7 :
		*x=48;
		*y=32;
		*x_center=142.5;
		*y_center=101.5;
		break;
	case 0 :
		*x=0;
		*y=48;
		*x_center=19.5;
		*y_center=142.5;
		break;
	case 1 :
		*x=16;
		*y=48;
		*x_center=60.5;
		*y_center=142.5;
		break;
	case 2 :
		*x=32;
		*y=48;
		*x_center=101.5;
		*y_center=142.5;
		break;
	case 3 :
		*x=48;
		*y=48;
		*x_center=142.5;
		*y_center=142.5;
		break;
	default:
		printf("Invalid module id\n");
		break;
	}
}
void getminmax(int moduleno,int*x,int *y,float* x_center,float* y_center)
{
	switch(moduleno)
	{
	case 12 :
		*x=0;
		*y=0;
		*x_center=19.5;
		*y_center=19.5;
		break;
	case 13 :
		*x=16;
		*y=0;
		*x_center=60.5;
		*y_center=19.5;
		break;
	case 14 :
		*x=32;
		*y=0;
		*x_center=101.5;
		*y_center=19.5;
		break;
	case 15 :
		*x=48;
		*y=0;
		*x_center=142.5;
		*y_center=19.5;
		break;
	case 8:
		*x=0;
		*y=16;
		*x_center=19.5;
		*y_center=60.5;
		break;
	case 9 :
		*x=16;
		*y=16;
		*x_center=60.5;
		*y_center=60.5;
		break;
	case 10 :
		*x=32;
		*y=16;
		*x_center=101.5;
		*y_center=60.5;
		break;
	case 11 :
		*x=48;
		*y=16;
		*x_center=142.5;
		*y_center=60.5;
		break;
	case 4 :
		*x=0;
		*y=32;
		*x_center=19.5;
		*y_center=101.5;
		break;
	case 5 :
		*x=16;
		*y=32;
		*x_center=60.5;
		*y_center=101.5;
		break;
	case 6 :
		*x=32;
		*y=32;
		*x_center=101.5;
		*y_center=101.5;
		break;
	case 7 :
		*x=48;
		*y=32;
		*x_center=142.5;
		*y_center=101.5;
		break;
	case 0 :
		*x=0;
		*y=48;
		*x_center=19.5;
		*y_center=142.5;
		break;
	case 1 :
		*x=16;
		*y=48;
		*x_center=60.5;
		*y_center=142.5;
		break;
	case 2 :
		*x=32;
		*y=48;
		*x_center=101.5;
		*y_center=142.5;
		break;
	case 3 :
		*x=48;
		*y=48;
		*x_center=142.5;
		*y_center=142.5;
		break;
	default:
		printf("Invalid module id\n");
		break;
	}
}
void getModule(int moduleNo,int *pixels)
{


	int i=0,j=0,ii=0,jj=0,temp=0;
	int cols=COLSPERMODULE;
	char moduleFileName[100]="module",blockageFileName[100]="bars_",tempBuff[100];
	FILE *fp,*blockage;
	int *data;
	char *CZTHOME = getenv ("CZTWORKSPACE");
	if(CZTHOME==NULL)
	{
		printf("CZTHOME Variable is not set\n");
		exit(0);
	}
	data=(int*)malloc(sizeof(int)*16*16);
	sprintf(tempBuff,"%d",moduleNo);
	strcpy(moduleFileName,CZTHOME);
	strcpy(blockageFileName,CZTHOME);
	strcat(moduleFileName,"/config/module");
	strcat(blockageFileName,"/config/bars_");
	strcat(moduleFileName,tempBuff);
	strcat(blockageFileName,tempBuff);
	fp=fopen(moduleFileName,"r");
	blockage=fopen(blockageFileName,"r");

	if(fp==NULL)
	{
		printf("Sorry Error while opening the module file\n%s\n",moduleFileName);
		exit(0);
	}
	if(blockage==NULL)
	{
		printf("Error while opening the blockage file\n%s\n",blockageFileName);
		exit(0);
	}
	int colPreVal=0,colInc=0,rowPreVal=0,rowInc=0;
	for(i=0;i<16;i++)
	{
		if(i==0||i==15)
		{
			rowInc=114;
		}
		else
		{
			rowInc=123;
		}

		for(j=0,colPreVal=0;j<16;j++)
		{
			if(j==0||j==15)
			{
				colInc=114;
			}
			else
			{
				colInc=123;
			}
			fscanf(fp,"%d",&data[i*16+j]);
			for(ii=rowPreVal;ii<rowPreVal+rowInc-10;ii++)
			{
				for(jj=colPreVal;jj<colPreVal+colInc;jj++)
				{

					pixels[ii*cols+jj]=data[i*16+j];
				}

			}
			temp=0;
			if(i!=15)
			{
				fscanf(blockage,"%d",&temp);
				for(ii=rowPreVal+rowInc-10;ii<rowPreVal+rowInc;ii++)
				{
					for(jj=colPreVal;jj<colPreVal+colInc;jj++)
					{
						pixels[ii*cols+jj]=temp;
					}
				}

			}
			else
			{
				for(ii=rowPreVal+rowInc-10;ii<rowPreVal+rowInc;ii++)
				{
					for(jj=colPreVal;jj<colPreVal+colInc;jj++)
					{
						pixels[ii*cols+jj]=data[i*16+j];
					}
				}


			}
			colPreVal+=colInc;
		}
		rowPreVal+=rowInc;
	}

}
void calculateSigmaClippedMean(float* pixel_count,float *mean_out,float*rms_out)
{
	int i=0;
	float mean_sum=0,rms_sum=0,mean,rms;
	int rows=NUMPIXPERMODULE,cols=NUMPIXPERMODULE;
	float temp_mean=0,temp_rms=0,temp_mean_sum=0,temp_rms_sum=0,mean_count=0,rms_count=0;
	mean_count=0;
	for(i=0;i<rows*cols;i++)
	{
		if(pixel_count[i]!=0)
		{
			mean_sum+=pixel_count[i];
			mean_count++;
		}
	}
	mean=mean_sum/(mean_count);
	rms_count=0;
	for(i=0;i<rows*cols;i++)
	{
		if(pixel_count[i]!=0)
		{
			rms_sum+=((pixel_count[i]-mean)*(pixel_count[i]-mean));
			rms_count++;
		}
	}
	rms=(rms_sum/rms_count);
	rms=sqrt(rms);

	while(1)
	{
		temp_mean_sum=0;
		temp_rms_sum=0;
		temp_mean=0;
		temp_rms=0;
		mean_count=0;
		rms_count=0;

		for(i=0;i<rows*cols;i++)
		{
			if((pixel_count[i])<((THREASHOLD*rms)+mean) || (pixel_count[i] > (mean-(THREASHOLD*rms))))
			{
				temp_mean_sum+=pixel_count[i];
				mean_count++;
			}
		}
		temp_mean=temp_mean_sum/mean_count;
		for(i=0;i<rows*cols;i++)
		{
			if((pixel_count[i])<((THREASHOLD*rms)+mean) || (pixel_count[i] > (mean-(THREASHOLD*rms))))
			{
				temp_rms_sum+=(pixel_count[i]-temp_mean)*(pixel_count[i]-temp_mean);
				rms_count++;
			}
		}
		if(mean_count!=rms_count)
			printf("Mean Rms count different\n");
		temp_rms=temp_rms_sum/rms_count;
		temp_rms=sqrt(temp_rms);

		float t1=0,t2=0;
		t1=((mean-temp_mean)/mean);
		t2=((rms-temp_rms)/rms);
		if(t1<0)
			t1*=-1;
		if(t2<0)
			t2*=-1;
		if(t1 <0.01 &&  t2<0.01 )
		{
			//mean=temp_mean;
			//rms=temp_rms;
			*mean_out=temp_mean;
			*rms_out=temp_rms;
			break;
		}
		else
		{

			mean=temp_mean;
			rms=temp_rms;
		}

	}
}
