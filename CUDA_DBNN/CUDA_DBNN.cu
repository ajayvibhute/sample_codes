#include "hip/hip_runtime.h"
/* CUDA version of the DBNN code for classification of stars, galaxies. The code was originally written by Prof. Sajeeth

Author: Ajay Vibhute

*/


#include <stdio.h>
#include <math.h>
#include <string.h>
#include <iostream>
using namespace std;
#include <stdlib.h>
#include<sys/times.h> // times() fun. is here.
#include <time.h>
#include <vector>
#define classes 500
#define max_resol 1600
#define features 100
#include"kernel.cu"
using std::vector;
/**************************
#include <stdlib.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/wait.h>

#include "vsipc.h"
***************************/

//#define oneround 100       //  Memory size.
//#define fst_gain 1.0 Moved to be a floating variable
static float bgain,gain,dmyclass[classes+2],classval[classes+2],cmax,c2max,c3max,c4max,tmp2_wts,totprob,oldj;
static float LoC=0.65;
static float nLoC=0.0;
static int jx=0, resol=100,nresol=0,nerror=0,nLoCcnt=1,skpchk=0,MissingDat=-9999;
static float omax,omin,rslt,rslt2,orslt,orslt2,prslt,nrslt,fst_gain;
clock_t start,stop;
static int argfnd, oneround=100,kmax,k2max,k3max,k4max,ans1,tcnt,rnn,rnd,i,j,k,l,n,m,p,c1cnt,c2cnt,pcnt,pocnt,invcnt,innodes=100,outnodes=100;
char fln[256],fltmp[256],urchoice,urchoicex,bcchoice,savedpar;
FILE *fl1,*fl2,*fl3,*fl4,*fl5,*fl6,*fl7,*fl8,*fl9,*fl10;
int main(int argv, char *argp[256])
/*
 Important note for revision in ver. 4.02
 Compute the bin center of gravities and the Kernel fit that holds the probability to
 find them. The slope of it should be used as bgain for each bin.
*/
/*
 You can now run dbnn in automated mode by specifying the parameters in 0.par and 1.par
 files. Also dbnn can now use the bin values from the saved apf file.
*/
{
	if(argv > 3)
	{
		argfnd=1;
		cout << "The selected option is " << *argp[3] <<"\n";    
		switch(*argp[3])
		{
			case '0':
				ans1=0;
				if((fl2=fopen("0.par","r"))!=NULL)
				{
					fscanf(fl2,"%c\n",&bcchoice);  //Handle missing or out of range values? Y if yes. NEW in Ver 7
					fscanf(fl2,"%c\n",&urchoice);
					fscanf(fl2,"%c\n",&savedpar);
					fscanf(fl2,"%c\n",&urchoicex);
					if(bcchoice == 'Y'||bcchoice =='y')
					{
						fscanf(fl2,"%d\n",&skpchk);
						if(skpchk <0) MissingDat=skpchk;
						cout << "System  is configured for handling missing data with missing data indicator" << MissingDat <<"\n";
					}
					fclose(fl2);			
				}
			else
			{ 
				cout << "No Parameter File... existing..";
				exit(1);
			}
			break;
			case '1':
				ans1=1;
				if((fl2=fopen("0.par","r"))!=NULL)
				{
					fscanf(fl2,"%c\n",&bcchoice);  //Handle missing or out of range values? Y if yes. NEW in Ver 7
					fscanf(fl2,"%c\n",&urchoice);
					fscanf(fl2,"%c\n",&savedpar);
					fscanf(fl2,"%c\n",&urchoicex);
					if(bcchoice == 'Y'||bcchoice =='y')
					{
						fscanf(fl2,"%d\n",&skpchk);
						if(skpchk <0) MissingDat=skpchk;
						cout << "System  is configured for handling missing data with missing data indicator" << MissingDat <<"\n";
					}
					fclose(fl2);			
				}
				else
				{ 
					cout << "No Parameter File... existing..";
					exit(1);
				}
				if((fl2=fopen("1.par","r"))!=NULL)
				{
					fscanf(fl2,"%f",&gain);
					fscanf(fl2,"%d",&oneround);
					fclose(fl2);			
				}
				else
				{ 
					cout << "No Parameter File... existing..";
					exit(1);
				}
			break;
			case '2':
				ans1=2;
				if((fl2=fopen("0.par","r"))!=NULL)
				{
					fscanf(fl2,"%c\n",&bcchoice);  //Handle missing or out of range values? Y if yes. NEW in Ver 7
					fscanf(fl2,"%c\n",&urchoice);
					fscanf(fl2,"%c\n",&savedpar);
					fscanf(fl2,"%c\n",&urchoicex);
					if(bcchoice == 'Y'||bcchoice =='y')
					{
						fscanf(fl2,"%d\n",&skpchk);
						if(skpchk <0) MissingDat=skpchk;
						cout << "System  is configured for handling missing data with missing data indicator" << MissingDat <<"\n";
					}
					fclose(fl2);			
				}
				else
				{ 
					cout << "No Parameter File... existing..";
					exit(1);
				}
			break;
			case '3':
				ans1=3;
				if((fl2=fopen("0.par","r"))!=NULL)
				{
					fscanf(fl2,"%c\n",&bcchoice);  //Handle missing or out of range values? Y if yes. NEW in Ver 7
					fscanf(fl2,"%c\n",&urchoice);
					fscanf(fl2,"%c\n",&savedpar);
					fscanf(fl2,"%c\n",&urchoicex);
					if(bcchoice == 'Y'||bcchoice =='y')
					{
						fscanf(fl2,"%d\n",&skpchk);
						if(skpchk <0) MissingDat=skpchk;
						cout << "System  is configured for handling missing data with missing data indicator" << MissingDat <<"\n";
					}
					fclose(fl2);			
				}
				else
				{ 
					cout << "No Parameter File... existing..";
					exit(1);
				}
			break;
				default:
					cout << "Create the APF file(0) or Create the Weights file (1) or Classify Data(2,3) ?";
					cin >> ans1;
			break;
		}
    }
    else
    {
		argfnd=0;
		cout << "Create the APF file(0) or Create the Weights file (1) or Classify Data(2,3) ?";
		cin >> ans1;
    }
    if(ans1 == 2)
    {
		if(argfnd==1)
		bgain=0.0;
		else
		{
			cout << "Allowed relaxation on the boundary (in % use 0 for default from training data) :";
			cin >> bgain;
			bgain=bgain*1.0;
		}
    }
    else
    bgain= 0;  // During training we are strict on boundary constraints.
    if(argv < 3)
    {
		cout << "Enter the name of the input file without extension (dat) :";
		cin >> fln;
    }
    else
    {
		strcpy(fln,argp[1]);
    }
    strcpy(fltmp,fln);
    strcat(fltmp,".dat");
/*
  The structure of the data file is:
  Feature1 Feature2 Feature3 ....(etc upto innodes) ActualClass
  Feature1 Feature2 Feature3 ....(etc upto innodes) ActualClass
  Feature1 Feature2 Feature3 ....(etc upto innodes) ActualClass
  The delimiters are spaces and not tabs!!
  ActualClass should be a numeric > 0
*/
    if((fl1=fopen(fltmp,"r"))!=NULL)
    {
		strcpy(fltmp,fln);
	    strcat(fltmp,".inf");
/*
  The format of the info file is: (in each line enter)
  innodes
  outnodes
  margin   <- This addition is required for regression problems.
  1.0       <- You can give any real positive value here. It is just a label.
  2.0
  ... (etc. upto no of classes)
  0.65 <- The Margin or Line of Control for marginal values.
  100 <- By default, the maximum bin size is set to 100. You can change this if required.
  0,1,2 <- no error bars, uniform error bar, upper lower separate error values per entry.
*/	
     	if((fl2=fopen(fltmp,"r"))!=NULL)
	    {
			i=0;
			fscanf(fl2,"%d",&innodes);
			fscanf(fl2,"%d",&outnodes);
			for (i=0;i<=outnodes;i++) // dmyclass[0] contains margin others are expected values.
			fscanf(fl2,"%f",&dmyclass[i]);
			fscanf(fl2,"%f",&LoC);   // New parameter to specify the Line Of Control
			fscanf(fl2,"%d",&nresol);
			fscanf(fl2,"%d",&nerror);
			cout <<"You have "<< innodes << " input nodes and " << outnodes <<" Output nodes with " << "margin set to " << LoC << " and error levels set to "<< nerror <<"\n";
			cout << "The target outputs are\n";
			for (i=0;i<=outnodes;i++) cout << dmyclass[i] <<"\n";
			if(nresol >0) 
			{
				resol=nresol;cout << "The maximum binsize is: " << resol <<"\n";
			}	
			else
			{
				cout << "The maximum binsize is: " << resol<<"\n";
			}
			fst_gain*=1.0/outnodes;
	    }
	    else
	    {
			cout << "Unable to find the Info file. Exiting !!";
			exit(1);
	    }
  
    } // program ends.
    else   // data file read error.
    {
		cout << "Unable to open the data file";
		exit(1);
    }
    cout << "Going to initialise the arrays\n";
 /**************** Let us Define the Network Structure *********************************/
//float mask_disp_maxres; // Space to save max resol for normalisation of mask_dist
    strcpy(fltmp,fln);
    strcat(fltmp,".dat");

	int numlines=getNumlines(fltmp);
	printf("NUMLINES:%d\n",numlines);

	float vectso[innodes+outnodes+2],tmpv,max[innodes+2],min[innodes+2],vects[innodes+outnodes+2];
	float err1vects[innodes+2], err2vects[innodes+2];
	//float arr_vects[numlines][innodes+outnodes+2];
	float *arr_tmpv=(float*)malloc(sizeof(float)*(numlines+2));
	float *arr_vects=(float*)malloc(sizeof(float)*(numlines+2)*(innodes+outnodes+2));
	float *arr_err1vects=(float*)malloc(sizeof(float)*(numlines+2)*(innodes+2));
	float *arr_err2vects=(float*)malloc(sizeof(float)*(numlines+2)*(innodes+2));
	int totsize=(innodes+2)*(resol+2)*(innodes+2)*(resol+4)*(outnodes+2);
	int totsendreceivesize=(innodes+1)*(resol+2)*(innodes+1)*(resol+1)*(outnodes+1);
	float *arr_anti_wts=(float*) malloc(totsize*sizeof(float));
	int *arr_anti_net=(int*)malloc(sizeof(int)*totsize);
	int ik=innodes+1,jk=resol+1,lk=innodes+1,mk=resol+1,kk=outnodes+1;
	int resolution[innodes+8];
	float classtot[innodes+2][resol+2];           // Total Prob. computed
	if(classtot==NULL){cout << "Out of Memory to Run Code at classtot.. Exiting\n";exit(1);}
	//float binloc[innodes+4][resol+8];
	float *arr_binloc=(float*)malloc(sizeof(float)*(innodes+4)*(resol+8));
	int rn=resol+1;
	int iin=innodes+4;





  /***************************Let us put up the Network***********************************/
//    Start the counter for case 2 here.................
	start = times(NULL);
	if (ans1==0)
	{
		n=0;
	    omax=-400;
	    omin=400;
	    while (!feof(fl1))
		{
			skpchk=0;
			for(i=1;i<=innodes;i++)
			if (n==0)
			{
			fscanf(fl1,"%f",&vects[i]); 
			if(nerror ==2){fscanf(fl1,"%f",&err1vects[i]);fscanf(fl1,"%f",&err2vects[i]);}else
			if(nerror ==1){fscanf(fl1,"%f",&err1vects[i]); err2vects[i]=err1vects[i];} 
			if(vects[i] != MissingDat)
				{
					min[i]=vects[i];
					max[i]=vects[i];
				}
				else max[i]=MissingDat;
			}
			else
			{
				fscanf(fl1,"%f",&vects[i]);
				if(vects[i] != MissingDat)
				{
					if( vects[i]> max[i]) max[i]=vects[i];
					if (min[i] > vects[i]) min[i]=vects[i];
				}
			}
			fscanf(fl1,"%f\n",&tmpv);
			if(tmpv>omax) omax = tmpv;
			if(tmpv<omin) omin =tmpv;
			k=1;
			j=1;
			n++;
		}
		cout << "No of vectors =" << n <<" and i/n is= " << 1.0/n << "\n";
		for(i=1;i<=innodes;i++)
		{
			if(min[i]==max[i])if(min[i]!=0){min[i]= -1.0*max[i];}else{min[i]=0.0; max[i]=1.0;}
		}
		if(argfnd==0)
		{
			cout <<"Do you want to use the saved parameters (Y/N)? ";
			cin >>savedpar;
		}
		if (savedpar == 'y') savedpar='Y';
		else
		if(savedpar == 'n') savedpar='N';
		if((savedpar == 'Y') || (savedpar=='y'))
		{
			strcpy(fltmp,fln);
			strcat(fltmp,".apf");
			fl2=NULL;
			if((fl2=fopen(fltmp,"r"))!=NULL)
			{
				cout << "Reading from the saved information\n";
				for (i=1;i<=innodes;i++)
				{
					fscanf(fl2,"%d",&resolution[i]);
					for(j=0;j<=resolution[i];j++) arr_binloc[(i*rn)+(j+1)]=j*1.0;
				}
				cout << innodes << " items read from " << fltmp <<"\n";
			}
			else
			{
				cout << "ERROR: File " << fltmp << " not found" << "\n";
				exit(1);
			}
		}
		else
		for(i=1;i<=innodes;i++)
		{
			if(min[i]==max[i])if(min[i]!=0){min[i]= -1.0*max[i];}else{min[i]=0.0; max[i]=1.0;}
			cin >> resolution[i];
			for(j=0;j<=resolution[i];j++) arr_binloc[(i*rn)+(j+1)]=j*1.0;

		}
		for(k=1;k<=outnodes;k++)
		for(i=1;i<=innodes;i++)
		for(j=0;j<=resolution[i];j++)
		for(l=1;l<=innodes;l++)
		for(m=0;m<=resolution[l];m++)
		{
			//anti_net[i][j][l][m][k]=1;
			//anti_wts[i][j][l][m][k]=(float)(1.0);
			arr_anti_wts[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]=(double)(1.0);
			arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]=1;
		}
      // Start the counter now...............
		start = times(NULL);
		rewind(fl1);
tcnt=0;
		while (!feof(fl1))
		{
			tcnt++;
            		for (i=1;i<=innodes;i++) 
            		{
				fscanf(fl1,"%f",&vects[i]); 
				if(nerror ==2){fscanf(fl1,"%f",&err1vects[i]);fscanf(fl1,"%f",&err2vects[i]);}else
				if(nerror ==1){fscanf(fl1,"%f",&err1vects[i]);err2vects[i]=err1vects[i];} 
			}
			fscanf(fl1,"%f\n",&tmpv);
			for(i=1;i<=innodes;i++)
			{
				if((vects[i] != MissingDat)&&(max[i] !=MissingDat))
				{
					vectso[i]=vects[i];
					vects[i]=round((vects[i]-min[i])/(max[i]-min[i])*resolution[i]);
					err1vects[i]=round((err1vects[i])/(max[i]-min[i])*resolution[i]);
					err2vects[i]=round((err2vects[i])/(max[i]-min[i])*resolution[i]);
				}
			}
			for (i=1;i<=innodes;i++)
			{
				j=0;
				if(vects[i] != MissingDat)
				{
//					oldj=(float)2*resolution[i];

					while ((fabs(vects[i]-arr_binloc[(i*rn)+(j+1)]) >=1.0 )&& (j<= resolution[i]))
					{
	//					oldj=fabs(vects[i]-binloc[i][j+1]);
						j++;
					}
					for (l=1;l<=innodes;l++)
					{
						m=0;
						if(i!=l)
						{
//							oldj=(float)2*resolution[l];
							while ((fabs(vects[l]-arr_binloc[(l*rn)+(m+1)]) >=1.0)&& (m<= resolution[l]))
							{
	//							oldj=fabs(vects[l]-binloc[l][m+1]);
								m++;
							}
							k=1;
							while ((k<=outnodes)&&(fabs(tmpv - dmyclass[k])) > dmyclass[0]) k++;
							//(anti_net[i][j][l][m][k])++;
							//(anti_net[i][j][l][m][0])++;
							(arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)])++;
							(arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+0)])++;


						}

					}
				}
			}
		}//end of while
		fclose(fl1);
		fclose(fl2);
		stop = times(NULL);
		cout << "The computation took " << fabs(start - stop)*10000/(CLOCKS_PER_SEC) << " Secs.\n";
         /*
            The conditional Probability,
	    P(A|B) = P(A intersection B)/P(B) is the
	    probability for the occurance of A(k) if B(ij) has happened =
	    Share of B(ij) that is held by A(k) / Probability of total B(ij)
	    in that particular feature i with resolution j.

                      */
		strcpy(fltmp,fln);
		strcat(fltmp,".awf");      // This file holds the weights
		fl6=fopen(fltmp,"w+");
		strcpy(fltmp,fln);
		strcat(fltmp,".apf");     // This file holds the estimated probability
		if((fl1=fopen(fltmp,"w+"))!=NULL)
		{
			for(i=1;i<=innodes;i++) fprintf(fl1,"%d ",resolution[i]);
			fprintf(fl1,"\n%f %f \n",omax,omin);
			for(i=1;i<=innodes;i++) fprintf(fl1,"%f ",max[i]);
			fprintf(fl1,"\n");
			for(i=1;i<=innodes;i++) fprintf(fl1,"%f ",min[i]);
			fprintf(fl1,"\n");
			for(k=1;k<=outnodes;k++)
			{
				for(i=1;i<=innodes;i++)
				for(j=0;j<=resolution[i];j++)
				{
					for(l=1;l<=innodes;l++)
					if(i!=l)
					{
						for(m=0;m<=resolution[l];m++)
						{
							//fprintf(fl1,"%d ",anti_net[i][j][l][m][k]);
							//fprintf(fl6,"%f ",(float)anti_wts[i][j][l][m][k]);

							fprintf(fl1,"%d ",arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]);
							fprintf(fl6,"%f ",(float)arr_anti_wts[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]);
						}
						fprintf(fl6,"\n");
						fprintf(fl1,"\n");
					}
				}
				fprintf(fl6,"\n");
				fprintf(fl1,"\n");
			}
			fprintf(fl6,"\n");
			fprintf(fl1,"\n");
		}
		else
		{
			cout << "Unable to create file for output\n";
			exit(1);
		}
		for(i=1;i<=innodes;i++)
		for(j=1;j<=resolution[i];j++)
		fprintf(fl6,"%f\n", (float)arr_binloc[(i*rn)+(j)]);                 /// Let us print the bins.
		fclose(fl1);
		fclose(fl6);
		fflush(NULL);
		cout << "Creating the Anticipated Weights data file\n";
	}
/**********************************End of Case 0 ******************************/












	if(ans1==1)
	{
		start = times(NULL);
		pcnt=0;
		pocnt=0;
		rslt=0.0;
		rslt2=0.0;
		orslt=rslt;
		orslt2=rslt2;

		for(i=0;i<totsize;i++)
			arr_anti_wts[i]=0;
		cout << "The programe will now modify the compensatory weights\n";
		if(argfnd==0)
		{
			cout << "Please enter the gain:";
			cin >> gain;
			cout << "Please enter the number of training epochs:";
			cin >> oneround;
		}
		// Start the counter in this round here...................
		start = times(NULL);
		strcpy(fltmp,fln);
		strcat(fltmp,".awf");
		if((fl6=fopen(fltmp,"r"))!=NULL)
		{
			strcpy(fltmp,fln);
			strcat(fltmp,".apf");
			fl2=NULL;
			if((fl2=fopen(fltmp,"r"))!=NULL)
			{
				for (i=1;i<=innodes;i++)
				{
					fscanf(fl2,"%d",&resolution[i]);
					for(j=0;j<=resolution[i];j++) arr_binloc[(i*rn)+(j+1)]=j*1.0;
				}

				fscanf(fl2,"\n%f",&omax);
				fscanf(fl2,"%f",&omin);
				fscanf(fl2,"\n");
				for(i=1;i<=innodes;i++) fscanf(fl2,"%f",&max[i]);
				fscanf(fl2,"\n");
				for(i=1;i<=innodes;i++) fscanf(fl2,"%f",&min[i]);
				fscanf(fl2,"\n");
				for(i=1;i<=innodes;i++)for(j=0;j<=resolution[i];j++)
				//for(l=1;l<=innodes;l++)for(m=0;m<=resolution[l];m++) anti_net[i][j][l][m][0] =0;
				for(l=1;l<=innodes;l++)for(m=0;m<=resolution[l];m++) arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+0)] =0;
				int ijk=0;
				for(k=1;k<=outnodes;k++)
				{
					for(i=1;i<=innodes;i++)
					for(j=0;j<=resolution[i];j++)
					{
						for(l=1;l<=innodes;l++)
						if(i!=l)
						{
							for(m=0;m<=resolution[l];m++)
							{
								ijk++;
								//fscanf(fl2,"%d",&anti_net[i][j][l][m][k]);
								//anti_net[i][j][l][m][0]+=anti_net[i][j][l][m][k];
								//fscanf(fl6,"%f",&anti_wts[i][j][l][m][k]);
								fscanf(fl2,"%d",&arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]);
								arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+0)]+=arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)];
								fscanf(fl6,"%f",&arr_anti_wts[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]);
							}
							fscanf(fl2,"\n");
							fscanf(fl6,"\n");
						}
					}
					fscanf(fl2,"\n");
					fscanf(fl6,"\n");
				}

				for(i=1;i<=innodes;i++)
				for(j=1;j<=resolution[i];j++)
					fscanf(fl6,"%f\n", &arr_binloc[(i*rn)+(j)]);                 /// Let us print the bins.

			}
			else
			{
				cout << "Unable to Open the APF information file\n";
				exit(1);
			}
			fclose(fl2);
		}
		else
		{
			cout << "Unable to Open the AWF information file\n";
			exit(1);
		}
		fclose(fl6);
		/*GPU Memory allocation*/

		int *d_arr_anti_net,*d_resolution;
		float *d_arr_anti_wts,*d_arr_tmpv,*d_arr_vects,*d_arr_err1vects,*d_arr_err2vects,*d_min,*d_max,*d_arr_binloc,*d_dmyclass;
		float *d_rslt,*d_rslt2,*tmp_rslt,*tmp_rslt2;
		int *d_pcnt,*tmp_pcnt,chunksize=0;
		hipError_t status ;
		tmp_pcnt=(int*)malloc(sizeof(int)*numlines);
		tmp_rslt=(float*)malloc(sizeof(float)*numlines);
		tmp_rslt2=(float*)malloc(sizeof(float)*numlines);
		for(i=0;i<numlines;i++)
			tmp_pcnt[i]=0;
		
		//allocate memory on GPU
		status=hipMalloc((void **)&d_pcnt,sizeof(int)*(numlines+10));
		status=hipMemset(d_pcnt, 0, sizeof(int)*(numlines+10));
		status=hipMalloc((void **)&d_rslt,sizeof(float)*(numlines+10));
		status=hipMemset(d_rslt, 0, sizeof(float)*(numlines+10));
		status=hipMalloc((void **)&d_rslt2,sizeof(float)*(numlines+10));
		status=hipMemset(d_rslt2, 0, sizeof(float)*(numlines+10));

		status=hipMalloc((void **)&d_arr_anti_net,sizeof(int)*totsize);
		status=hipMemset(d_arr_anti_net, 0, sizeof(int)*totsize);
		status=hipMalloc((void **)&d_resolution,sizeof(int)*(innodes+8));
		status=hipMemset(d_resolution, 0, sizeof(int)*(innodes+8));

		status=hipMalloc((void **)&d_arr_anti_wts,sizeof(float)*totsize);
		status=hipMemset(d_arr_anti_wts, 0, sizeof(int)*totsize);

		status=hipMalloc((void **)&d_arr_tmpv,sizeof(float)*(numlines+2));
		status=hipMemset(d_arr_tmpv, 0, sizeof(float)*(numlines+2));

		status=hipMalloc((void **)&d_arr_vects,sizeof(float)*(numlines+2)*(innodes+outnodes+2));
		status=hipMemset(d_arr_vects, 0, sizeof(float)*(numlines+2)*(innodes+outnodes+2));

		status=hipMalloc((void **)&d_arr_err1vects,sizeof(float)*(numlines+2)*(innodes+2));
		status=hipMemset(d_arr_err1vects, 0, sizeof(float)*(numlines+2)*(innodes+2));

		status=hipMalloc((void **)&d_arr_err2vects,sizeof(float)*(numlines+2)*(innodes+2));
		status=hipMemset(d_arr_err2vects, 0, sizeof(float)*(numlines+2)*(innodes+2));

		status=hipMalloc((void **)&d_min,sizeof(float)*(innodes+2));
		status=hipMemset(d_min, 0, sizeof(float)*(innodes+2));

		status=hipMalloc((void **)&d_max,sizeof(float)*(innodes+2));
		status=hipMemset(d_max, 0,sizeof(float)*(innodes+2));

		status=hipMalloc((void **)&d_arr_binloc,sizeof(float)*(innodes+4)*(resol+8));
		status=hipMemset(d_arr_binloc, 0, sizeof(float)*(innodes+4)*(resol+8));

		status=hipMalloc((void **)&d_dmyclass,sizeof(float)*(classes+2));
		status=hipMemset(d_dmyclass, 0, sizeof(float)*(classes+2));

		if (status != hipSuccess)
			printf("Error in cuda memory allocation\n");


		for(rnd=0;rnd<=oneround;rnd++)     // Training round starts here....
		{
			if((n==pocnt)&& (n>0)){ printf("breaking\n"); break;}
			strcpy(fltmp,fln);
			strcat(fltmp,".dat");
			fl1=fopen(fltmp,"r");
			n=0;
			rslt=0.0;
			rslt2=0.0;
			pcnt=0;
			int cindex=0;
			for(cindex=0;cindex<numlines;cindex++)
			{
				for(k=1;k<=outnodes;k++) classval[k]=1.0;
				n++;
				if(ans1==3)
				{
					for (i=1;i<=innodes;i++)
					{
						fscanf(fl1,"%f",&arr_vects[(cindex*innodes)+i]);
						if(nerror ==2){fscanf(fl1,"%f",&arr_err1vects[(cindex*innodes)+i]);fscanf(fl1,"%f",&err2vects[(cindex*innodes)+i]);}else
						if(nerror ==1){fscanf(fl1,"%f",&arr_err1vects[(cindex*innodes)+i]);err2vects[(cindex*innodes)+i]=arr_err1vects[(cindex*innodes)+i];}
					}
					fscanf(fl1,"\n");
				}
				else
				{
					for (i=1;i<=innodes;i++)
					{
						fscanf(fl1,"%f",&arr_vects[(cindex*innodes)+i]);
						if(nerror ==2){fscanf(fl1,"%f",&arr_err1vects[(cindex*innodes)+i]);fscanf(fl1,"%f",&arr_err2vects[(cindex*innodes)+i]);}else
						if(nerror ==1){fscanf(fl1,"%f",&arr_err1vects[(cindex*innodes)+i]);arr_err2vects[(cindex*innodes)+i]=arr_err1vects[(cindex*innodes)+i];}
					}
					fscanf(fl1,"%f\n",&arr_tmpv[cindex]);
				}
			}
			fclose(fl1);

			hipMemcpy(d_arr_anti_net,arr_anti_net,sizeof(int)*totsize,hipMemcpyHostToDevice);
			hipMemcpy(d_resolution,resolution,sizeof(int)*(innodes+8),hipMemcpyHostToDevice);
			hipMemcpy(d_arr_anti_wts,arr_anti_wts,sizeof(float)*totsize,hipMemcpyHostToDevice);
			hipMemcpy(d_arr_tmpv,arr_tmpv,sizeof(float)*(numlines+2),hipMemcpyHostToDevice);
			hipMemcpy(d_arr_vects,arr_vects,sizeof(float)*(numlines+2)*(innodes+outnodes+2),hipMemcpyHostToDevice);
			hipMemcpy(d_arr_err1vects,arr_err1vects,sizeof(float)*(numlines+2)*(innodes+2),hipMemcpyHostToDevice);
			hipMemcpy(d_arr_err2vects,arr_err2vects,sizeof(float)*(numlines+2)*(innodes+2),hipMemcpyHostToDevice);
			hipMemcpy(d_min,min,sizeof(float)*(innodes+2),hipMemcpyHostToDevice);
			hipMemcpy(d_max,max,sizeof(float)*(innodes+2),hipMemcpyHostToDevice);
			hipMemcpy(d_arr_binloc,arr_binloc,sizeof(float)*(innodes+4)*(resol+8),hipMemcpyHostToDevice);
			hipMemcpy(d_dmyclass,dmyclass,sizeof(float)*(classes+2),hipMemcpyHostToDevice);


			int numblocks=ceil(numlines/512.0);
			kernel1(numlines,arr_tmpv,min,max,resolution,arr_vects,arr_err1vects,arr_err2vects,arr_binloc,rn,arr_anti_net,arr_anti_wts,dmyclass,gain,innodes,resol,outnodes,nerror,rnd);

			strcpy(fltmp,fln);
			strcat(fltmp,".dat");
			fl1=fopen(fltmp,"r");

			m=n;
			n=0;
			rslt=0.0;
			rslt2=0.0;
			pcnt=0;


//			while (!feof(fl1))                    // Test round...
			for(cindex=0;cindex<numlines;cindex++)
			{
				n++;

				if(ans1==3)
				{
					for (i=1;i<=innodes;i++)
					{
						fscanf(fl1,"%f",&arr_vects[(cindex*innodes)+i]);
						if(nerror ==2){fscanf(fl1,"%f",&arr_err1vects[(cindex*innodes)+i]);fscanf(fl1,"%f",&arr_err2vects[(cindex*innodes)+i]);}else
						if(nerror ==1){fscanf(fl1,"%f",&arr_err1vects[(cindex*innodes)+i]);arr_err2vects[(cindex*innodes)+i]=arr_err1vects[(cindex*innodes)+i];}
					}
					fscanf(fl1,"\n");
				}
				else
				{
					for (i=1;i<=innodes;i++)
					{
						fscanf(fl1,"%f",&arr_vects[(cindex*innodes)+i]);
						if(nerror ==2){fscanf(fl1,"%f",&arr_err1vects[(cindex*innodes)+i]);fscanf(fl1,"%f",&arr_err2vects[(cindex*innodes)+i]);}else
						if(nerror ==1){fscanf(fl1,"%f",&arr_err1vects[(cindex*innodes)+i]);arr_err2vects[(cindex*innodes)+i]=arr_err1vects[(cindex*innodes)+i];}
					}
					fscanf(fl1,"%f\n",&arr_tmpv[cindex]);
				}

			}
			fclose(fl1);
			i=0;

			status=hipMemset(d_pcnt, 0, sizeof(int)*(numlines+10));
			status=hipMemset(d_rslt, 0, sizeof(float)*(numlines+10));
			status=hipMemset(d_rslt2, 0, sizeof(float)*(numlines+10));
			//copy the results
			hipMemcpy(d_resolution,resolution,sizeof(int)*(innodes+8),hipMemcpyHostToDevice);
			hipMemcpy(d_arr_tmpv,arr_tmpv,sizeof(float)*(numlines+2),hipMemcpyHostToDevice);
			hipMemcpy(d_arr_vects,arr_vects,sizeof(float)*(numlines+2)*(innodes+outnodes+2),hipMemcpyHostToDevice);
			hipMemcpy(d_arr_err1vects,arr_err1vects,sizeof(float)*(numlines+2)*(innodes+2),hipMemcpyHostToDevice);
			hipMemcpy(d_arr_err2vects,arr_err2vects,sizeof(float)*(numlines+2)*(innodes+2),hipMemcpyHostToDevice);
			hipMemcpy(d_min,min,sizeof(float)*(innodes+2),hipMemcpyHostToDevice);
			hipMemcpy(d_max,max,sizeof(float)*(innodes+2),hipMemcpyHostToDevice);
			hipMemcpy(d_arr_binloc,arr_binloc,sizeof(float)*(innodes+4)*(resol+8),hipMemcpyHostToDevice);
			hipMemcpy(d_dmyclass,dmyclass,sizeof(float)*(classes+2),hipMemcpyHostToDevice);

			if((status = hipGetLastError()) != hipSuccess)
			{
				printf("Error(%s:%d) %s\n",__FILE__,__LINE__,hipGetErrorString(status));
			}

			numblocks=ceil(numlines/512.0);
			kernel2<<<numblocks,512>>>(numlines,d_arr_tmpv,d_min,d_max,d_resolution,d_arr_vects,d_arr_err1vects,d_arr_err2vects,d_arr_binloc,rn,d_arr_anti_net,d_arr_anti_wts,d_dmyclass,gain,innodes,resol,outnodes,nerror,rnd,d_pcnt,d_rslt,d_rslt2);


			if((status = hipGetLastError()) != hipSuccess)
			{
				printf("%s\n",hipGetErrorString(status));
			}
			hipMemcpy(tmp_pcnt,d_pcnt,sizeof(int)*numlines,hipMemcpyDeviceToHost);
			if((status = hipGetLastError()) != hipSuccess)
			{
				printf("%s\n",hipGetErrorString(status));
			}
			hipMemcpy(tmp_rslt,d_rslt,sizeof(float)*numlines,hipMemcpyDeviceToHost);
			if((status = hipGetLastError()) != hipSuccess)
			{
				printf("%s\n",hipGetErrorString(status));
			}
			hipMemcpy(tmp_rslt2,d_rslt2,sizeof(float)*numlines,hipMemcpyDeviceToHost);
			if((status = hipGetLastError()) != hipSuccess)
			{
				printf("%s\n",hipGetErrorString(status));
			}

			pcnt=0;
			rslt=0;
			rslt2=0;
			for(i=0;i<numlines;i++)
			{
				pcnt+=tmp_pcnt[i];
				rslt+=tmp_rslt[i];
				rslt2+=tmp_rslt2[i];
			}



			printf("rnd:%d\trslt:%f\tRslt2:%f\tOrslt2:%f\tpcnt:%d\n",rnd,rslt,rslt2,orslt2,pcnt);

			kmax=1;
			if(orslt2==0) orslt2=rslt2;
			if(orslt==0) orslt=rslt;

			prslt=(rslt2-orslt2);
			if(rslt > 0)
			nrslt=(orslt/rslt);
			if(pcnt>pocnt)
			{
				rnn=rnd;
				pocnt=pcnt;   // The best result is now saved in pocnt
				strcpy(fltmp,fln);
				strcat(fltmp,".awf");
				fl6=fopen(fltmp,"w+");
				kmax=1;
				for(k=1;k<=outnodes;k++)
				{
					for(i=1;i<=innodes;i++)
					for(j=0;j<=resolution[i];j++)
					{
						for(l=1;l<=innodes;l++)
						if(i!=l)
						{
							for(m=0;m<=resolution[l];m++)
							{
								//fprintf(fl6,"%f ",anti_wts[i][j][l][m][k]);
								fprintf(fl6,"%f ",arr_anti_wts[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]);

							}
							fprintf(fl6,"\n");
						}
					}
					fprintf(fl6,"\n");
				}
				fprintf(fl6,"\n");
				for(i=1;i<=innodes;i++)
				for(j=1;j<=resolution[i];j++)
				fprintf(fl6,"%f\n", arr_binloc[(i*rn)+(j)]);                 /// Let us print the bins.
				fflush(fl6);
				fclose(fl6);
				cout << "Round:" << rnn << "| TProb["<<prslt<<"," <<nrslt<<"] | Passed count:" << pocnt << endl;
				if(orslt2 <rslt2) orslt2=rslt2;
				if(rslt < orslt) orslt=rslt;
			}
			n=m;
		}  //rnd inc.
		fl6=NULL;
		cout << "Best result at round " << rnn<< endl;
	}  // ans <> 1
/***********************************End of Case 1*******************************/







    strcpy(fltmp,fln);
    strcat(fltmp,".dat");
    fl1=fopen(fltmp,"r");
    strcpy(fltmp,fln);
    strcat(fltmp,".awf");
    fl6=NULL;
    fl6=fopen(fltmp,"r");
    strcpy(fltmp,fln);
    strcat(fltmp,".apf");
    fl2=NULL;
    if((fl2=fopen(fltmp,"r"))!=NULL)
	{
		cout << "Creating the Anticipated Network outputs\n";
		for (i=1;i<=innodes;i++)
		{ 
			fscanf(fl2,"%d",&resolution[i]);
			for(j=0;j<=resolution[i];j++) arr_binloc[(i*rn)+(j+1)]=j*1.0;
		}
		fscanf(fl2,"%f",&omax);
		fscanf(fl2,"%f",&omin);
		fscanf(fl2,"\n");
        for(i=1;i<=innodes;i++) fscanf(fl2,"%f",&max[i]);
        fscanf(fl2,"\n");
    	for(i=1;i<=innodes;i++) fscanf(fl2,"%f",&min[i]);
		fscanf(fl2,"\n");
		for(i=1;i<=innodes;i++)for(j=0;j<=resolution[i];j++) 
		for(l=1;l<=innodes;l++)for(m=0;m<=resolution[l];m++) arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+0)] =0;
        for(k=1;k<=outnodes;k++)
        {
			for(i=1;i<=innodes;i++)
			for(j=0;j<=resolution[i];j++)
			{
				for(l=1;l<=innodes;l++)
				if(i!=l)
				{
					for(m=0;m<=resolution[l];m++)
					{

						fscanf(fl2,"%d",&arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]);
						fscanf(fl6,"%f",&arr_anti_wts[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]);
						arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+0)]+=(float)(arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]);
					}
					fscanf(fl2,"\n");
					fscanf(fl6,"\n");
				}
			}
			fscanf(fl2,"\n");
			fscanf(fl6,"\n");
		}
    }
    else
    {
		cout << "Unable to Open the APF information file";
		exit(1);
    }
    for(i=1;i<=innodes;i++)
    for(j=1;j<=resolution[i];j++)
    {
		fscanf(fl6,"%f\n",&arr_binloc[(i*rn)+(j)]);                 /// Let us print the bins.
    }
    fclose(fl6);
    fl4=fopen("output.dat","w+");  // Network Output values
    cout << "Read all input parameters\n";
// *********** case 3 ***********************************************
    if (ans1 !=3)
    {
		fl5=fopen("actual.dat","w+");  // Expected Output Values
		strcpy(fltmp,fln);
		strcat(fltmp,argp[2]);
		strcpy(fltmp,fln);
		strcat(fltmp,argp[2]);
		strcat(fltmp,".cmp");         // Lets see how well the classification went.
		fl7=fopen(fltmp,"w+");
		fprintf(fl7,"Sample         Predicted     Actual            Prediction \n");
		fprintf(fl7," No.       Ist 2nd  3rd  4th  item             Confidence\n");
		c1cnt=0;
		c2cnt=0;
		invcnt=0;
		n=0;
    }
 // Create classtot values ***********************
    while (!feof(fl1))
	{
		n++;
		cmax= 0.0;
		c2max=0.0;
		c3max=0.0;
		c4max=0.0;
		kmax=0;
		k2max=0;
		k3max=0;
		k4max=0;
        classval[0]=0.0;
	    if(ans1==3)
	    {
			for (i=1;i<=innodes;i++) 
			{
				fscanf(fl1,"%f",&vects[i]);
				if(nerror ==2){fscanf(fl1,"%f",&err1vects[i]);fscanf(fl1,"%f",&err2vects[i]);}else
				if(nerror ==1){fscanf(fl1,"%f",&err1vects[i]);err2vects[i]=err1vects[i];} 
			}
			fscanf(fl1,"\n");
	    }
	    else
	    {
			for (i=1;i<=innodes;i++) 
			{
				fscanf(fl1,"%f",&vects[i]);
				if(nerror ==2){fscanf(fl1,"%f",&err1vects[i]);fscanf(fl1,"%f",&err2vects[i]);}else
				if(nerror ==1){fscanf(fl1,"%f",&err1vects[i]);err2vects[i]=err1vects[i];} 
			}
			fscanf(fl1,"%f\n",&tmpv);
	    }
        skpchk=0;
	    for(i=1;i<=innodes;i++)
        {
			vectso[i]=vects[i]; 
            if((((max[i]-min[i]) >0)&& (vects[i] !=MissingDat))&&(max[i] !=MissingDat)) 
            {
				vects[i]=round(((vects[i]-min[i])/(max[i]-min[i]))*resolution[i]);
				err1vects[i]=round((err1vects[i])/(max[i]-min[i])*resolution[i]);
				err2vects[i]=round((err2vects[i])/(max[i]-min[i])*resolution[i]);
				skpchk=0;
		    }
            else
            skpchk=1;
        }
		for(k=1;k<=outnodes;k++) classval[k]=1.0; tmp2_wts=1.0;
 		for (i=1;i<=innodes;i++)
		{
			j=0;
            if(vects[i]==MissingDat)
				skpchk=1;
            else
				skpchk=0;
            if ((resolution[i] >= vects[i]) &&(skpchk==0))
            {
                while ((fabs(vects[i]-arr_binloc[(i*rn)+(j+1)]) >=1.0)&& (j<= resolution[i]))
                {
                    j++;
                }
                jx=0;
            }
			else
			{
//NSP_added jx=-1;				  
			  jx=1;
			}
	        for (l=1;l<=innodes;l++)
	        {
				if((i!=l) && (jx==0))
				{
					m=0;
					if((vects[l]==MissingDat)||(vects[i]==MissingDat))
						skpchk=1;
					else
						skpchk=0;
					if ((resolution[l] >= vects[l]) &&(skpchk==0))
					{
						while ((fabs(vects[l]-arr_binloc[(l*rn)+(m+1)]) >=1.0)&& (m<= resolution[l]))
						{       
							m++;
						}
					}
					for (k=1;k<=outnodes;k++)
					{

						if(jx==0){tmp2_wts=(float)arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)];}else{tmp2_wts=1.0/outnodes;}
						if(nerror ==2) 
						{
							for(p=(m-(int)err1vects[l]);p<=(m+(int)err2vects[l]);p++)
							{

								if(p<0) p=0; if(p>resolution[l]) break;
								if ((float)arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(p*kk)+k)] > tmp2_wts)
								m=p;
							}
						}
						if(nerror ==1) 
						{
							for(p=(m-(int)err1vects[l]);p<=(m+(int)err1vects[l]);p++)
							{

								if(p<0) p=0; if(p>resolution[l]) break;
								if ((float)arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(p*kk)+k)] > tmp2_wts)
								m=p;
							}
						}

						if((arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+0)] > 0) && (resolution[i]>= vects[i])&& (resolution[l]>= vects[l])&&(skpchk==0))
						{

							if(jx==0){tmp2_wts=(float)arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]*arr_anti_wts[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]*1.0/(arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+0)]);}
							else{tmp2_wts=1.0/outnodes;}
						}
						else
						if(skpchk == 1)   // || bcchoice == 'y')
						{
							tmp2_wts= 1.0; //(float)1.0/outnodes; //1.0; //
						}
						else
						{
							tmp2_wts=(float)1.0/outnodes;
						}
						if((resolution[i] >= vects[i])&& (resolution[l]>= vects[l])&&(skpchk==0))
						{
							classval[k]*=(float)tmp2_wts;
						}
					}
					totprob=0;
					for(k=1;k<=outnodes;k++) totprob+=classval[k];
					if (totprob==0) {totprob=innodes*outnodes; cout <<"Caution!! Item did not have known types\n";}
					for(k=1;k<=outnodes;k++) classval[k]=classval[k]/totprob;
 				
				}
			}
	    }
       cmax=0.0;
        c2max=0.0;
        c3max=0.0;
        k3max=0.0;
        kmax=0.0;
        k2max=0.0;
	    totprob=0.0;
	    for (k=1;k<=outnodes;k++)
	    {
			if (classval[k] > cmax)
	        {
				c4max=c3max;
				k4max=k3max;
				c3max=c2max;
				k3max=k2max;
                c2max=classval[kmax];
	            k2max=kmax;
	            cmax=classval[k];
	            kmax=k;
            }
            else
            if (classval[k]>c2max)
            {
				c4max=c3max;
				k4max=k3max;
				c3max=c2max;
				k3max=k2max;
                c2max=classval[k];
	            k2max=k;
            }
           else
            if (classval[k]>c3max)
            {
				c4max=c3max;
				k4max=k3max;
                c3max=classval[k];
	            k4max=k;
            }
           else
            if (classval[k]>c4max)
            {
                c4max=classval[k];
	            k4max=k;
            }
	        totprob += (float)classval[k];
        }
	    if(totprob <=0.0) totprob=innodes*outnodes;
        if(ans1 ==3)
		{
			if (dmyclass[(int)kmax]- (int)dmyclass[(int)kmax] ==0.0)
			{
				fprintf(fl4,"%d  %d %-5.2f %d %-5.2f %d %-5.2f %d %-5.2f",n, (int)dmyclass[(int)kmax],100.0*((classval[kmax])/totprob),(int)dmyclass[(int)k2max],100.0*((classval[k2max])/totprob),(int)dmyclass[(int)k3max],100.0*((classval[k3max])/totprob),(int)dmyclass[(int)k4max],100.0*((classval[k4max])/totprob));
		    }
		    else
		    {
				fprintf(fl4,"%d  %f %-5.2f %f %-5.2f %f %-5.2f %f %-5.2f",n, dmyclass[(int)kmax],100.0*((classval[kmax])/totprob),dmyclass[(int)k2max],100.0*((classval[k2max])/totprob),dmyclass[(int)k3max],100.0*((classval[k3max])/totprob),dmyclass[(int)k4max],100.0*((classval[k4max])/totprob));
            }
			if((fabs(classval[kmax]-classval[k2max]))<0.01*classval[kmax]) //classval[kmax])
			{
				nLoC+=classval[kmax]/totprob;
	            nLoCcnt++;
				if(classval[kmax]>totprob*LoC)    //LoC)
				{
					fprintf(fl4, " <-- Either of it"); 
				}
				else
				{
					fprintf(fl4, " <-- Rejected");
				}
			}
			else
			{
				if(classval[kmax]>totprob*LoC)    //LoC)
				{
					fprintf(fl4, " <-- confident");
				}
				else
				{
					fprintf(fl4, " <-- Rejected");
				}
			}
			fprintf(fl4,"\n");
		}
		if(ans1 !=3)
		{
			if (dmyclass[(int)kmax]- (int)dmyclass[(int)kmax] ==0.0)
			{
				fprintf(fl4,"%d  %d\n",n, (int)dmyclass[(int)kmax]);
				fprintf(fl7, "%-8d    %d   %d     %d  %d     %d   ",n,(int)dmyclass[(int)kmax],(int)dmyclass[(int)k2max],(int)dmyclass[(int)k3max],(int)dmyclass[(int)k4max],(int)tmpv);
		    }
		    else
		    {
				fprintf(fl4,"%d  %f\n",n, dmyclass[(int)kmax]);
				fprintf(fl7, "%-8d    %f   %f     %f    %f     %f    ",n,dmyclass[(int)kmax],dmyclass[(int)k2max],dmyclass[(int)k3max],dmyclass[(int)k4max],tmpv);
	        }
			if(fabs(dmyclass[kmax]-tmpv) >= dmyclass[0])
			{
				if (classval[kmax]==0.0)
				{
					invcnt++;
                    fprintf(fl7, "%-5.2f %% <-Out of range %-5.2f %% \n",100.0*((classval[kmax])/totprob),100.0*((classval[k2max])/totprob));
				}
				else
				{
					if (fabs(dmyclass[k2max]-tmpv) < dmyclass[0])
					{
						if((fabs(classval[kmax]-classval[k2max]))<0.01*classval[k2max]) //classval[kmax])
						{
							nLoC+=classval[kmax]/totprob;
							nLoCcnt++;
							if (classval[kmax]>totprob*LoC) // LoC)
							{
								c2cnt++;  // No more differences. NSP (OCT 2001)
								fprintf(fl7, "%-5.2f %% <-F(1)P(2) %-5.2f %%  %-5.2f %% %-5.2f %% \n",100.0*((classval[kmax])/totprob),100.0*((classval[k2max])/totprob),100.0*((classval[k3max])/totprob),100.0*((classval[k4max])/totprob));
							}
							else
							{
								fprintf(fl7, "%-5.2f %%  <-FMC %-5.2f %% %-5.2f %% %-5.2f %% \n",100.0*((classval[kmax])/totprob),100.0*((classval[k2max])/totprob),100.0*((classval[k3max])/totprob),100.0*((classval[k4max])/totprob));
								invcnt++;
							}
						}
						else
						{
							if (classval[kmax]>totprob*LoC) // LoC)
							{
								fprintf(fl7, "%-5.2f %% <-Failed %-5.2f %%  %-5.2f %%  %-5.2f %% \n",100.0*((classval[kmax])/totprob),100.0*((classval[k2max])/totprob),100.0*((classval[k3max])/totprob),100.0*((classval[k4max])/totprob));
							}
							else
							{
								fprintf(fl7, "%-5.2f %% <-FMC %-5.2f %%  %-5.2f %%  %-5.2f %% \n",100.0*((classval[kmax])/totprob),100.0*((classval[k2max])/totprob),100.0*((classval[k3max])/totprob),100.0*((classval[k4max])/totprob));
								invcnt++;
							}
						}
					}
					else
					{
						if (classval[kmax]>totprob*LoC) // LoC)
						{
							fprintf(fl7, "%-5.2f %% <-Failed %-5.2f %%  %-5.2f %% %-5.2f %% \n",100.0*((classval[kmax])/totprob),100.0*((classval[k2max])/totprob),100.0*((classval[k3max])/totprob),100.0*((classval[k4max])/totprob));
						}
						else
						{
							fprintf(fl7, "%-5.2f %% <-FMC %-5.2f %% %-5.2f %% %-5.2f %% \n",100.0*((classval[kmax])/totprob),100.0*((classval[k2max])/totprob),100.0*((classval[k3max])/totprob),100.0*((classval[k4max])/totprob));
							invcnt++;
						}
					}
				}
			}
			else
			{
				if((fabs(classval[kmax]-classval[k2max]))<0.01*classval[kmax])
				{
					nLoC+=classval[kmax]/totprob;
					nLoCcnt++;
					if (classval[kmax]>totprob*LoC) // LoC)
					{
						fprintf(fl7, "%-5.2f %% <-P(1)F(2) %-5.2f %% %-5.2f %% %-5.2f %% \n",100.0*((classval[kmax])/totprob),100.0*((classval[k2max])/totprob),100.0*((classval[k3max])/totprob),100.0*((classval[k4max])/totprob));
						c1cnt++;
					}
					else
					{
						invcnt++;
						fprintf(fl7, "%-5.2f %% <-PMC %-5.2f %% %-5.2f %% %-5.2f %% \n",100.0*((classval[kmax])/totprob),100.0*((classval[k2max])/totprob),100.0*((classval[k3max])/totprob),100.0*((classval[k4max])/totprob));
					}
				}
				else
				{ 
					if (classval[kmax]>totprob*LoC) // LoC)
					{
						fprintf(fl7, "%-5.2f %% <-Passed %-5.2f %% %-5.2f %% %-5.2f %% \n",100.0*((classval[kmax])/totprob),100.0*((classval[k2max])/totprob),100.0*((classval[k3max])/totprob),100.0*((classval[k4max])/totprob));
						c1cnt++;
					}
					else
					{
						invcnt++;
						fprintf(fl7, "%-5.2f %% <-PMC %-5.2f %% %-5.2f %% %-5.2f %% \n",100.0*((classval[kmax])/totprob),100.0*((classval[k2max])/totprob),100.0*((classval[k3max])/totprob),100.0*((classval[k4max])/totprob));
					}
				}
			}
			fprintf(fl5,"%d %e \n",n,(float) tmpv);
		} // ans1 != 3 ends here ******************
	}
	cout << "The suggested LoC is " << nLoC/nLoCcnt << "\n";
	fclose(fl1);
	fclose(fl2);
	fclose(fl4);
	if(ans1 < 3)
	{
		strcpy(fltmp,fln);
//	   tmp2_wts=0.0;
		fclose(fl5);
		fprintf(fl7,"*________________________________________________________________________\n");
		fprintf(fl7,"*Total    Success in   Success in   Non classified   Real success in    \n");
		cout << "*________________________________________________________________________\n";
		cout << "*Total    Success in   Success in   Non classified   Real success in    \n";
		if (outnodes > 3)
	    {
			fprintf(fl7,"* No.    Ist Choice  2nd Choice     items           two chances    \n");
     		fprintf(fl7,"* %d       %d          %d           %d             %-5.2f %% \n",n,c1cnt,c2cnt,invcnt,(float)100.0*(c1cnt+c2cnt)/(n-invcnt));
     		cout << "* No.    Ist Choice  2nd Choice     items           two chances    \n";
     		printf("* %d       %d          %d           %d             %-5.2f %% \n",n,c1cnt,c2cnt,invcnt,(float)100.0*(c1cnt+c2cnt)/(n-invcnt));
     	}
     	else
     	{
			fprintf(fl7,"* No.    Ist Choice  2nd Choice     items           First chance    \n");
     		fprintf(fl7,"* %d       %d          %d           %d             %-5.2f %% \n",n,c1cnt,c2cnt,invcnt,(float)100.0*(c1cnt)/(n-invcnt));
     		cout << "* No.    Ist Choice  2nd Choice     items           First chance    \n";
     		printf("* %d       %d          %d           %d             %-5.2f %% \n",n,c1cnt,c2cnt,invcnt,(float)100.0*(c1cnt)/(n-invcnt));
     	}
	  	fprintf(fl7,"*________________________________________________________________________\n");
		printf("*________________________________________________________________________\n");
		fclose(fl7);
	} // ******** ans1!=3 ends here *************
	cout << "Done.\n";
	stop = times(NULL);
	cout << "The computation took " << fabs(start - stop)*10000/(CLOCKS_PER_SEC) << " Secs.\n";
} //end main



