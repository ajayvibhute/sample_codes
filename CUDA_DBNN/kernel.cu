#include "hip/hip_runtime.h"
/*
 * kernel.cu
 *
 *  Created on: May 22, 2015
 *  Author: Ajay Vibhute
 */




void kernel1(int numlines,float *arr_tmpv,float *min,float *max,int* resolution,float *arr_vects,float *arr_err1vects,float *arr_err2vects,float*arr_binloc,int rn,int *arr_anti_net,float *arr_anti_wts,float*dmyclass, float gain,int innodes,int resol,int outnodes,int nerror,int rnd)
{
	int cindex=0,i,j,k,l,m,p,jx=0,MissingDat=-9999,kmax=0;
	float tmp2_wts=0,tmpv=0,totprob=0,cmax=0;
	int jk=resol+1,lk=innodes+1,mk=resol+1,kk=outnodes+1;
	float classval[classes+2];
	for(cindex=0;cindex<numlines;cindex++)
	{
		tmpv=arr_tmpv[cindex];
		for(i=1;i<=innodes;i++)
		{
			if((arr_vects[(cindex*innodes)+i] != MissingDat)&&(max[i]!=MissingDat))
			{
				arr_vects[(cindex*innodes)+i]=round((arr_vects[(cindex*innodes)+i]-min[i])/(max[i]-min[i])*resolution[i]);
				arr_err1vects[(cindex*innodes)+i]=round((arr_err1vects[(cindex*innodes)+i])/(max[i]-min[i])*resolution[i]);
				arr_err2vects[(cindex*innodes)+i]=round((arr_err2vects[(cindex*innodes)+i])/(max[i]-min[i])*resolution[i]);
			}
		}
		for(k=1;k<=outnodes;k++) classval[k]=1.0;
		for (i=1;i<=innodes;i++)
		{
			j=0;
			k=1;
			if(arr_vects[(cindex*innodes)+i] != MissingDat)
			{

				while ((fabs(arr_vects[(cindex*innodes)+i]-arr_binloc[(i*rn)+(j+1)]) >=1.0)&& (j<= resolution[i]))
				{
					j++;
				}
				//NSP_added if(fabs(vects[i]-binloc[i][j+1]) <= binloc[i][0]){jx=0;} else{jx=-1;}
				if(fabs(arr_vects[(cindex*innodes)+i]-arr_binloc[(i*rn)+(j+1)]) <= 1.0){jx=0;} else{jx=-1;}

				for (l=1;l<=innodes;l++)
				{
				  if(i!=l)
				  {
					m=0;
					k=1;
					if(arr_vects[(cindex*innodes)+l] != MissingDat)
					{
						while ((fabs(arr_vects[(cindex*innodes)+l]-arr_binloc[(l*rn)+(m+1)]) >=1.0)&& (m<= resolution[l]))
						{
							m++;
						}
						for (k=1;k<=outnodes;k++)
						{

							if(jx==0)tmp2_wts=(float)arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)];
							else tmp2_wts=1.0/outnodes;
							if(nerror ==2)
							{
								for(p=(m-(int)arr_err1vects[(cindex*innodes)+l]);p<=(m+(int)arr_err1vects[(cindex*innodes)+l]);p++)
								{
									if(p<0) p=0; if(p>resolution[l]) break;

									if ((float)arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(p*kk)+k)] > tmp2_wts)
									m=p;
								}
							}
							if(nerror ==1)
							{
								for(p=(m-(int)arr_err1vects[(cindex*innodes)+l]);p<=(m+(int)arr_err1vects[(cindex*innodes)+l]);p++)
								{

									if(p<0) p=0; if(p>resolution[l]) break;
									if ((float)arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(p*kk)+k)] > tmp2_wts)
									m=p;
								}
							}

							if(arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+0)] > 0)
							{
								if(jx==0)
									tmp2_wts=(float)arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]*arr_anti_wts[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]/(arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+0)]);
								else
								tmp2_wts=1.0/outnodes;
							}
							else
								tmp2_wts= 1.0/outnodes;
							classval[k]*=(float)tmp2_wts;
						}
						totprob=0;
						for(k=1;k<=outnodes;k++) totprob+=classval[k];
						if (totprob==0) {totprob=innodes*outnodes; /*cout <<"Caution : Item has no representation type\n";*/}
						for(k=1;k<=outnodes;k++) classval[k]=classval[k]/totprob;
					 }
				  }
				}
			}
		}
		kmax=1;
		cmax=0;
		for (k=1;k<=outnodes;k++)
		{
			if (classval[k] > cmax)
			{
				cmax=classval[k];
				kmax=k;
			}
		}
		if ((fabs(dmyclass[kmax]-tmpv) >= dmyclass[0]) && (rnd >0))
		{
			for (i=1;i<=innodes;i++)
			{
				j=0;
				k=1;
				if(arr_vects[(cindex*innodes)+i] != MissingDat)
				{
					while ((fabs(arr_vects[(cindex*innodes)+i]-arr_binloc[(i*rn)+(j+1)]) >=1.0)&& (j<= resolution[i]))
					{
						j++;
					}
					for (l=1;l<=innodes;l++)
					{
					if(i!=l)
					{
						m=0;
						k=1;
						if(arr_vects[(cindex*innodes)+l] != MissingDat)
						{
							while ((fabs(arr_vects[(cindex*innodes)+l]-arr_binloc[(l*rn)+(m+1)]) >=1.0)&& (m<= resolution[l]))
							{
								m++;
							}
							while ((k<=outnodes)&&fabs(dmyclass[k]-tmpv) > dmyclass[0]) k++;
							if((classval[(int)kmax] >0)&&(classval[k]<classval[(int)kmax]))
							{

								arr_anti_wts[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]+=(float)gain*(1.0-(classval[k]/classval[(int)kmax]));
							}
							/*if(arr_anti_wts[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)] <= 0.0)
								cout << k << " "<< tmpv << "[" << dmyclass[1] << "]" << dmyclass[outnodes] << "\n";*/
						}
					}
					}
				}
			}
		} // kmax che
	} // while not eof check
}









__global__ void kernel2(int numlines,float *arr_tmpv,float *min,float *max,int* resolution,float *arr_vects,float *arr_err1vects,float *arr_err2vects,float*arr_binloc,int rn,int *arr_anti_net,float *arr_anti_wts,float*dmyclass,
		float gain,int innodes,int resol,int outnodes,int nerror,int rnd,int*pcnt,float*rslt,float*rslt2)
{

	int cindex=0,i,j,k,l,m,p,jx=0,MissingDat=-9999,kmax=0;
	float tmp2_wts=1,tmpv=0,totprob=0,cmax=0,oldj;
	int jk=resol+1,lk=innodes+1,mk=resol+1,kk=outnodes+1;
	float classval[classes];
	cindex=threadIdx.x +( blockIdx.x * blockDim.x);
	if(cindex<numlines)
	{

		kmax=1;
		cmax=0;
		tmpv=arr_tmpv[cindex];
		for(i=1;i<=innodes;i++)
		{
			if((arr_vects[(cindex*innodes)+i] != MissingDat)&&(max[i]!=MissingDat))
			{
				arr_vects[(cindex*innodes)+i]=round((arr_vects[(cindex*innodes)+i]-min[i])/(max[i]-min[i])*resolution[i]);
				arr_err1vects[(cindex*innodes)+i]=round((arr_err1vects[(cindex*innodes)+i])/(max[i]-min[i])*resolution[i]);
				arr_err2vects[(cindex*innodes)+i]=round((arr_err2vects[(cindex*innodes)+i])/(max[i]-min[i])*resolution[i]);
				if (arr_vects[(cindex*innodes)+i] < 0) arr_vects[(cindex*innodes)+i]=0;             // let us be bounded. #Oct 2001.
			}
		}
		for(k=1;k<=outnodes;k++) classval[k]=1.0;
		for (i=1;i<=innodes;i++)
		{
			j=0;
			k=1;
			if(arr_vects[(cindex*innodes)+i] != MissingDat)
			{
				while ((fabs(arr_vects[(cindex*innodes)+i]-arr_binloc[(i*rn)+(j+1)]) >=1.0)&& (j<= resolution[i]))
				{
					j++;
				}
				//NSP_added if(fabs(vects[i]-binloc[i][j+1]) <= binloc[i][0]){jx=0;} else{jx=-1;}
				if(fabs(arr_vects[(cindex*innodes)+i]-arr_binloc[(i*rn)+(j+1)]) < 1.0){jx=0;} else{jx=-1;}
				for (l=1;l<=innodes;l++)
				{
				 if (i !=l)
				 {
					m=0;
					k=1;
					if(arr_vects[(cindex*innodes)+l] != MissingDat)
					{
						oldj=(float)2*resolution[l];
						while ((fabs(arr_vects[(cindex*innodes)+l]-arr_binloc[(l*rn)+(m+1)]) >=1.0)&& (m<= resolution[l]))
						{
							oldj=fabs(arr_vects[(cindex*innodes)+l]-arr_binloc[(l*rn)+(m+1)]);
							m++;
						}
						for (k=1;k<=outnodes;k++)
						{

							if(jx==0)
							tmp2_wts=(float)arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)];
							else
							tmp2_wts=1.0/outnodes;
							if(nerror ==2)
							{
								for(p=(m-(int)arr_err1vects[(cindex*innodes)+l]);p<=(m+(int)arr_err2vects[(cindex*innodes)+l]);p++)
								{
									if(p<0) p=0; if(p>resolution[l]) break;

									if ((float)arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(p*kk)+k)] > tmp2_wts)
									m=p;
								}
							}
							if(nerror ==1)
							{
								for(p=(m-(int)arr_err1vects[(cindex*innodes)+l]);p<=(m+(int)arr_err1vects[(cindex*innodes)+l]);p++)
								{

									if(p<0) p=0; if(p>resolution[l]) break;
									if ((float)arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(0*kk)+k)] > tmp2_wts)
									m=p;
								}
							}

								if(arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+0)] > 0)
								{
									if(jx==0)
									tmp2_wts=(float)arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]*arr_anti_wts[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+k)]/(arr_anti_net[((i*jk*lk*mk*kk)+(j*lk*mk*kk)+(l*mk*kk)+(m*kk)+0)]);
									else
									tmp2_wts=1.0/outnodes;
								}
							else
								tmp2_wts=(float)1.0/outnodes;
							classval[k]*=(float)tmp2_wts;
						}

						totprob=0;
						for(k=1;k<=outnodes;k++) totprob+=classval[k];
						if (totprob==0) {totprob=innodes*outnodes; }
						for(k=1;k<=outnodes;k++) classval[k]=classval[k]/totprob;
					}
				 }
				}
			}
		}
		for (k=1;k<=outnodes;k++)
		{
			if (classval[k] > cmax)
			{
				cmax=classval[k];
				kmax=k;
			}
		}

		if (fabs(dmyclass[kmax]-tmpv) < dmyclass[0])
		{
			rslt2[cindex]+=cmax;
			pcnt[cindex]+=1;
		}
		else
		{
			k=1;
			while ((k<=outnodes)&&fabs(dmyclass[k]-tmpv) > dmyclass[0]) k++;
			rslt[cindex]+=(cmax-classval[k]);
		}

		} // while not eof check

}

int getNumlines(char*filename)
{
	FILE *fp=fopen(filename,"r");
	char ch;
	int lines=0;
	while(!feof(fp))
	{
	  ch = fgetc(fp);
	  if(ch == '\n')
	  {
	    lines++;
	  }
	}
	fclose(fp);
	return lines;


}
