#include "hip/hip_runtime.h"
/************************************************************************************************************************
 Simultaneous  Linear Least Square Fit via Singular Value Decomposition for Multiple Data sets

Technologies:C,CUDA.
Author:Ajay Vibhute
Project Guide:Prof.Dipankar Bhattacharya.
Project For: IUCAA
*************************************************************************************************************************/
//Header Files.
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<pthread.h>
#include<time.h>
#include<unistd.h>
#include "PreProcessors.h"
#include"generate_xy.h"
#include"svd_cuda.c"
extern "C"
void startExecution(char *observationId); 
#define NUM_THREADS 3

#define print(format,...) fprintf(format,__VA_ARGS__)

//function to initilise the cuda environment and map GPU to a MPI process
void cudaInit(int myrank)
{

	float *initmalloc=NULL;

	hipSetDevice(myrank);
	hipMalloc(&initmalloc,sizeof(float));
}

//This function reads the data and performs the forward fitting for a given sky range
void*svdfit(void * data)
{
	//creating object to store the source information 
	InfoData * info=(InfoData*)data;

	//creating local variable and initilising them

	char filename[BUFSIZE],command[BUFSIZE],observation_id[BUFSIZE];	
	char string[100];
	char fileName[BUFSIZE],xtemp[BUFSIZE],ytemp[BUFSIZE],stemp[BUFSIZE];

	int myrank=info->myrank;
	int xstart1=0,xend1=0,ystart1=0,yend1=0;
	int intensityCounter=0;
	int rv=0,jj;
	int trueCondition=1;
	int index=0,updateCounter=0;
	int threadId=myrank;
	int counter=0, no_another=0,*maskpat=NULL;
	int noOfGrids=noOfDataSet;
	int noOfbins=rows;
	int ielem=0, ipat=0;	
	int icam=myrank;//threadId;

	float *x=NULL,*a=NULL,*chisq=NULL,*x_another=NULL,*x_updated=NULL,*x_passing=NULL;
	float intensity=1,thetaX_Updated=0,thetaY_Updated=0.0;
	float *thetaX=NULL,*thetaY=NULL;
	float xdiff1=0.0,ydiff1=0.0;
	float tolerance=0.001;
	float xCheck=0.0,yCheck=0.0,tempChi=0.0;
	float timeSpent=0;

	FILE  *print_pointer=NULL;
	FILE *fp=NULL,*read_observed_shadow=NULL;
	FILE *pipe=NULL; 


	strcpy(observation_id,info->observationId);
	if(myrank<3)
	{
		cudaInit(myrank);

		//checking mode of the excution
		#ifdef __SIM__
	
			print_pointer=stdout;
	
		#else
			//populating the directory structure
			strcpy(filename,OBSERVED_SHADOW_PATH);
			strcat(filename,observation_id);
			strcat(filename,"/output");
			strcat(filename,"/");

			//populating the log file name. Log of individual camera will be stored in the different file
			if(myrank==0)
				strcat(filename,"BOOM_Execution_Log");
	
			else if(myrank==1)
				strcat(filename,"SLANT1_Execution_Log");
			else
				strcat(filename,"SLANT2_Execution_Log");

			//creating log file
			print_pointer=fopen(filename,"w");
	
			if(print_pointer==NULL)
			{
				print(stderr,"%s:%d:Error while creating output file %s \n",__FILE__,__LINE__,filename);
				printf("%s:%d:Error while creating output file %s \n",__FILE__,__LINE__,filename);	
				return (void*)-1;
			}
		#endif

		//initilizing the grid information
		xstart1 =info->xs;             //starting of thetaX
		xend1 =info->xe;              //end of the thetax
		ystart1=info->ys;            //starting of thetaY
		yend1 =info->ye;  
		xdiff1=xdiff;	
		ydiff1=ydiff;

		//getting size of the sky grid
		xgridSize=(int)((xend1-xstart1)/xdiff1);
		ygridSize=(int) ((yend1-ystart1)/ydiff1);
		ygridSize++;
		if(xdiff1>0.1)
			xgridSize++;
		noOfDataSet=xgridSize*ygridSize;
		thetaX=(float*)malloc(sizeof(float)*(xgridSize+1)*(ygridSize+1));
		thetaY=(float*)malloc(sizeof(float)*(xgridSize+1)*(ygridSize+1));
		//Running a loop over the grid and initilizing the pointing information
		for(float ty=ystart1;ty<=yend1;ty+=ydiff1)
		{	
			for(float tx=xstart1;tx<=xend1;tx+=xdiff1)
			{	
				thetaX[counter]=tx;
				thetaY[counter++]=ty;
			}
		}
		counter=0;

		no_another=0;	

		size_t size=sizeof(float)*(noOfbins+2)*noOfGrids;

		/* Allocaing host (CPU) memory  */
		x=(float*)malloc(size);
		x_updated=(float*)malloc(size);
		x_passing=(float*)malloc(size);
		a=(float*)malloc(size);
		chisq=(float*)malloc(size);
		x_another=(float*)malloc(size*no_another);
		maskpat=(int*)malloc(NUM_PATTERNS*(NUM_MASK_ELEM+2)*sizeof(int));

		//Reading Maskpattern from the file ./config/MASKPATTERNS/....
		for(ipat = 0; ipat <= NUM_PATTERNS-1; ++ipat)
		{
			sprintf(string, MASK_PATH_FORMAT, ipat+1);
		
			if((fp = fopen(string, "r")) == NULL)
			{
				print(stderr,"\nError: (%s:%d) Mask Pattern file \"%s\" could not be opened for reading\n", __FILE__, 	__LINE__, string);
			}
	
			//setting up the mask pattern
			maskpat[ipat*(NUM_MASK_ELEM+2)+0] = 0; /* pad closed elem in the beginning */
			maskpat[ipat*(NUM_MASK_ELEM+2)+(NUM_MASK_ELEM+1)] = 0;     /* and at the end */
		
			//reading the mask elements	
			for(ielem = 1; ielem <= NUM_MASK_ELEM; ++ielem)
				fscanf(fp, "%1d", &maskpat[ipat*(NUM_MASK_ELEM+2)+ielem]);
	
			//closing the file
			fclose(fp);
		}
		/*
		Generating the shadow for the given location with the given intensity
		when flag is set to 0 then new shadow is generated and return (void*)-1;
		when flag is set to 1 then new shadow is genereated and added to the previous shadow which is 
		in the buffer.
		
		If 5 or 6 info->sources out of 10 are at the edge then it will give nearly correct location of the all info->sources 
		if number of info->sources at the edge is increased beyound that then it will not work properly i.e., it will not 
                give the correct strength and correct location of the info->sources
		If info->sources are not at the edge of the grid and maximum source strength is 2000 and minimum source strength
 		is 10 then also it's able to detect all the info->sources with it's relative strength
		*/


	
		sprintf(fileName,OBSERVED_SHADOW_PATH);
		strcat(fileName,observation_id);
		//One telescope is assigned to one node and each node will run the processing on 3 GPUs
		if(myrank==0)
			strcat(fileName,"/BOOM");
		else if(myrank==1)
			strcat(fileName,"/SLANT1");
		else 
			strcat(fileName,"/SLANT2");

		//reading observation data/shadows
		read_observed_shadow=fopen(fileName,"r");
		if(read_observed_shadow==NULL)
		{
			print(print_pointer,"%s (%d):%s file not found\n",__FILE__ ,__LINE__,fileName);
			fclose(print_pointer);
			return (void*)-1;;
		}
		for( int k=1;k<=noOfbins;k++)
		{
			rv=fscanf(read_observed_shadow,"%f",&x[k]);
			if(rv==-1)
			{
				print(print_pointer,"%s (%d):%s file have insufficient data to proceed\n",__FILE__ ,__LINE__,fileName);
				fclose(print_pointer);
				return (void*)-1;;
			
			}
		}
		fclose(read_observed_shadow);
		

	
/************************************************************************************************************************
	Pass User defined Function of X
	x-another contains the values for another function of x noOfbins wise not column wise.
	If input for the x_another is read from the file then read all values of first function for all column then all 	values of the second function and so on... if you pass values column wise then it will give error or the output 	values would not be correct
************************************************************************************************************************/
	/*
	for(k=0;k<noOfGrids;k++)
	{
		for(i=1;i<=noOfbins;i++)
		{
			temp=no_another;	
			for(int l=0;l<no_another;l++)
			{
				x_another[counter++]=powf(x[k*noOfbins+i],colms-temp);	//change the counter++
				temp--;
			}
			
		}
		
		
	}
	*/
/************************************************************************************************************************

	Call SVD_FIT to find coordinates,chisquare and cvm matrix
	It provide two overloaded function one with user defined function and one without user defined function.
	1)timeSpent=SVD_FIT(x,y,noOfbins,colms,sig,a,chisq,cvm,noOfGrids,no_another,x_another);
	2)timeSpent=SVD_FIT(x,y,noOfbins,colms,sig,a,chisq,cvm,noOfGrids);
	Function will return (void*)-1; the time spent on GPU for exection 
*************************************************************************************************************************/

	info->numSources=0;

	//running a loop till we discover a source in a field of view. Sometimes a source may take multiple
	//iterations to get detected. In such cases windowing is performed and source intensity is populated accordingly.
	while(trueCondition)
	{
	
		updateCounter++;
		intensityCounter=0;
		for(int kk=1;kk<noOfbins;kk++)
		{
			x_passing[kk]=0;
			x_passing[kk]=x[kk]-(x_updated[kk]);	
		}
		//performing fitting on thee GPU
		timeSpent=SVD_FIT(x_passing,thetaX,thetaY,a,chisq,maskpat,icam);

		bzero(fileName,sizeof(fileName));
		strcpy(fileName,"../output/");
		sprintf(string,"%ld",threadId+1);
		strcat(fileName,string);
		strcat(fileName,"_output.txt");
		index=0;
		tempChi=chisq[0];	
		counter=1;
	
		//checking chisquare value of each grid point and computing lowest chisquare value
		for(int i=0;i<noOfGrids;i++)
		{
			tempChi=chisq[i];
			index=i;
			for(int j=0;j<noOfGrids;j++)
			{
				if(tempChi>chisq[j])
				{
					tempChi=chisq[j];
					index=j;	
					
				}
			
			}
			if(intensityCounter==0)
			{
				intensity=a[index*colms+2];
				thetaX_Updated=thetaX[index];
				thetaY_Updated=thetaY[index];
				intensityCounter++;
			
			}
		chisq[index]=10000;
		
		}
	
				
		//updating the source information
		if(info->numSources==0)
		{
			info->sources[0].x_position=thetaX_Updated;
			info->sources[0].y_position=thetaY_Updated;
			info->sources[0].sourceStrength=intensity;
			info->numSources++;
		}	
		else
		{
		

			
			for(jj=0;jj<info->numSources;jj++)
			{
				xCheck=info->sources[jj].x_position-thetaX_Updated;
				yCheck=info->sources[jj].y_position-thetaY_Updated;
				if(xCheck<0)
					xCheck*=-1;
				if(yCheck<0)
					yCheck*=-1;
				//updating source intensity if source is already listed
				if(xCheck<=(0.5*xdiff1) && yCheck<=(2.5) )	
				{
					info->sources[jj].sourceStrength+=intensity;	
					break;
				}
			}
			if(jj==info->numSources)
			{
				if(intensity>tolerance)
				{	
					if(intensity>0.1)
					{
						info->sources[info->numSources].x_position=thetaX_Updated;
						info->sources[info->numSources].y_position=thetaY_Updated;
						info->sources[info->numSources].sourceStrength=intensity;
						info->numSources++;
					}
					else	
					{
						trueCondition=0;
					}
				
				}
				else
				{	
					trueCondition=0;
				}
		
			}
		}
		//generating shadow for updated location	
		generateShadow(thetaX_Updated,thetaY_Updated,intensity,1,x_updated,icam);
		print(print_pointer,"...................%d th pass rank %d ..\n",updateCounter,myrank);
		print(print_pointer,"\tThetaX=%f\tThetaY=%f\n",thetaX_Updated,thetaY_Updated);
		print(print_pointer,"\tSource Strength=%f\n",intensity);

		}//end of updateCounter loop
		print(print_pointer,"\n-----------------------Processing Complete by thread %d -------------------\n",myrank);
	
		//Printing the results
		print(print_pointer,"NoOfGrids processed=%d\n",noOfGrids);
		print(print_pointer,"X Resolution=%f\tY Resolution=%f\n",xdiff1,ydiff1);
		print(print_pointer,"No of sources Found =%d\n",info->numSources);
		print(print_pointer,"sources with their position and strength\n\n");
		print(print_pointer,"\tThetaX\t\tThetaY\t\tsourcestrength\n\n");
		for(int kk=0;kk<info->numSources;kk++)
		{
			print(print_pointer,"\t%f \t %f \t %f\n",info->sources[kk].x_position,info->sources[kk].y_position,info->sources[kk].sourceStrength);
		}
	
		strcpy(filename,OBSERVED_SHADOW_PATH);
		strcat(filename,observation_id);
		strcat(filename,"/output");
		strcat(filename,"/");	
		if(myrank==0)
			strcat(filename,"BOOM_SOURCE_LIST");
		else if(myrank==1)
			strcat(filename,"SLANT1_SOURCE_LIST");
		else
		strcat(filename,"SLANT2_SOURCE_LIST");
			
		//creating a file and writing source information
		fp=fopen(filename,"w");
		if(fp==NULL)
		{
			print(print_pointer,"%s (%d):Error while creating file %s \n",__FILE__ ,__LINE__,fileName);
			return (void*)-1;;
		}
		for(int i=0;i<info->numSources;i++)
		{
			fprintf(fp,"\t%.2f",info->sources[i].x_position);
			fprintf(fp,"\t\t%.2f",info->sources[i].y_position);
			fprintf(fp,"\t\t%.2f\n",info->sources[i].sourceStrength);
		}
		fclose(fp);
	
		//Generating a plot indicating the source location in the field of view using GNU plot	

		pipe= popen("gnuplot -persist","w");
		strcpy(command,"set title 'Located sources By ");
		if(myrank==0)
			strcat(command," Boom Camera '\n");
		else if(myrank==1)
			strcat(command," SLANT1 Camera '\n");
		else
			strcat(command," SLANT2 Camera '\n");

		fprintf(pipe,command);
		fprintf(pipe, "set xlabel 'X'\n");
		fprintf(pipe, "set ylabel 'Y'\n");
		for(int i=0;i<info->numSources;i++)
		{
			strcpy(command,"set obj ");
			sprintf(xtemp,"%d",i+1);
			strcat(command,xtemp);
			strcat(command," circle  center ");
			sprintf(xtemp,"%.2f",info->sources[i].x_position);
			strcat(command,xtemp);
			strcat(command,",");
			sprintf(ytemp,"%.2f",info->sources[i].y_position);
			strcat(command,ytemp);
			strcat(command," radius 0.1 fill solid fc rgbcolor 'red'\n");
			fprintf(pipe,command);
			sprintf(stemp,"%0.2f",info->sources[i].sourceStrength);
			bzero(command,sizeof(command));
			strcpy(command,"set label ' ( ");
			strcat(command,xtemp);
			strcat(command,",");
			strcat(command,ytemp);
			strcat(command,",");
			strcat(command,stemp);
			strcat(command,")' at ");
			strcat(command,xtemp);
			strcat(command,",");
			strcat(command,ytemp);
			fprintf(pipe,"%s\n",command);
		}
		fprintf(pipe, "plot -40\n");
		fclose(pipe);

#ifdef __SIM__
#else
	fclose(print_pointer);
#endif

	//Free allocated memory
	free(x);
	free(chisq);	
	free(x_another);
	free(thetaX);
	free(thetaY);
	free(maskpat);
	freeDevice();	
		
}	
	return (void*) 0;
}//main

void startExecution(char *observationId)
{
	char hostname[100];
	gethostname(hostname,sizeof(hostname));
	printf("Executing %s on  node %s\n",observationId,hostname);	
	InfoData info[NUM_THREADS+1];	
	
	time_t t1,t2;	
	(void) time(&t1);
	pthread_t  thread[NUM_THREADS];
	for(int i=0;i<NUM_THREADS;i++)
	{
		info[i].myrank=i;
		info[i].numSources=i;
		if(i==0)
		{
			info[i].xs=-10;
			info[i].xe=10;
		}
		else
		{
			info[i].xs=-13;
			info[i].xe=13;
		}
		info[i].ys=-45;
		info[i].ye=45;
		strcpy(info[i].observationId,observationId);
		//printf("ystart=%d\tyend=%d\n",info[i].ys,info[i].ye);
		pthread_create(&thread[i],NULL,svdfit,&info[i]);
		sleep(5);
		
	}
	for(int i=0;i<NUM_THREADS;i++)
	{
		pthread_join(thread[i],NULL);
	}
	(void) time(&t2);
	
	//printf("Total time spent by all threads is=%f\n",(float)(t2-t1));
	printf("Execution of %s is complete on node %s\n",observationId,hostname);		
	return ;
	//pthread_exit(NULL);
	//printf("Exit");
	
}
