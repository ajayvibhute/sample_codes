#include "hip/hip_runtime.h"
/*

	This program generates the DPH for given input angle thetaX and thetaY
	Here the total no of photons accepted is 2000000
	This program uses GPU computing, To execute it for all the four quadrant use the command

	mpirun -np 3 -host cn001 ./cudaevent 2.3215 2.3215 : -np 1 -host cn002 ./cudaevent
	This will use three gpu cards from first node and one gpu card from second node.
	Author: Ajay Vibhute
*/
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>
#include<time.h>
#include<sys/time.h>
#include<hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "fitsio.h"
#include <mpi.h>
#define PI 3.14159265
#define noBlock 10
#define noThread 256
#define TOTALACCEPTED 25600

//declaring the required functions
void printerror( int );
__global__ void kernel(float tx,float ty,float height,int *maskPattern,float * dphValues,int *,int *,int,int,int*,int*,int*,int*,int);
__device__ void generateEvent(float tx,float ty,float height,int *maskPattern,float * dphValues,int *,int*,int,int*,int*,int*,int*,int);
void executeKernel(float tx,float ty,int myrank,int gpuId,int*accepted,int*rejected,int);
__device__ void getDetectorIdPixelNo(int x,int y,int *moduleNo,int *pixelNo,int detectorId);
void getTimeEnergy(char * filename,int *time,int * energy);
void cudaInit(int gpuId);

//start of main
int main(int argc,char*argv[])
{

	int myrank=0,npes=0;
	float timeSpent=0.0;
	fitsfile *fptr=NULL;
	int status=0;
    	char output_filename[100] ="",temp[100],hostname[100];    
    	int bitpix   =  FLOAT_IMG; /* 16-bit unsigned short pixel values       */
 	float tx=0,ty=0;
	int gpuId=0,totalAccepted=0,totalRejected=0,totalGeneratedCount=0,totalGeneratedCountMean=0;
	int *x=NULL,*y=NULL,*detectorId=NULL,*pixelNo=NULL, *time=NULL,*energy=NULL;

	MPI_Status stat;
	MPI_Init(&argc,&argv);
	MPI_Comm_size(MPI_COMM_WORLD,&npes);
	MPI_Comm_rank(MPI_COMM_WORLD,&myrank);	
	totalGeneratedCountMean++;
	totalGeneratedCount=TOTALACCEPTED;//2001920;
	//reading inputs at rank 0
	if(myrank==0)
	{
		int buf[TOTALACCEPTED];
		MPI_Buffer_attach( buf, TOTALACCEPTED+1000 );		
		timeSpent=MPI_Wtime();
				
		if(argv[1]==NULL||argv[2]==NULL)
		{
			printf("Enter value for ThetaX\n");
			scanf("%f",&tx);
			printf("Enter value for ThetaY\n");
			scanf("%f",&ty);
		}
		else
		{
			tx=(float)atof(argv[1]);
			ty=(float)atof(argv[2]);
		}
		
		/*
			Calculate the total generated Count by using poisson distribution.
		*/
		
	}
	MPI_Barrier(MPI_COMM_WORLD);
	
	//Broadcasting the inputs
	MPI_Bcast(&tx,1,MPI_FLOAT,0,MPI_COMM_WORLD);
	MPI_Bcast(&ty,1,MPI_FLOAT,0,MPI_COMM_WORLD);
	
	//Assigning GPU cards	
	if(myrank!=3)
	{
		gpuId=myrank;
	}
	else
	{
		//The last MPI process will run on another node, so first GPU card will be used.
		gpuId=0;
	}
	
	//calling kernal	
	executeKernel(tx,ty,myrank,gpuId,&totalAccepted,&totalRejected,totalGeneratedCount);

	gethostname(hostname,sizeof(hostname));
	if(myrank==0)
	{
		//writing event file

		time=(int*)malloc(sizeof(int)*totalGeneratedCount);
		energy=(int*)malloc(sizeof(int)*totalGeneratedCount);
		x=(int*)malloc(sizeof(int)*totalGeneratedCount);
		y=(int*)malloc(sizeof(int)*totalGeneratedCount);
		detectorId=(int*)malloc(sizeof(int)*totalGeneratedCount);
		pixelNo=(int*)malloc(sizeof(int)*totalGeneratedCount);
		int tfields   =6;      /* table will have 3 columns */
    		long nrows    = TOTALACCEPTED;       /* table will have 6 rows    */
		char extname[] = "EVENT";           /* extension name */
		char *ttype[] = { "TIME", "PHA", "DETID","PIXID","DETX","DETY" };
   		char *tform[] = { "1I",     "1I",       "1I"  ,"1I" ,"1I","1I" };
    		char *tunit[] = { "s",      "\0",       "\0"  , "\0" "\0","\0"};
		long firstrow=1, firstelem=1;
		status=0;	
		

		sprintf(temp,"%f",tx);
		strcat(output_filename,temp);
		strcat(output_filename,"_");
		bzero(temp,sizeof(temp));
		sprintf(temp,"%f",ty);
		strcat(output_filename,temp);
		strcat(output_filename,".event");
		bzero(temp,sizeof(temp));
		strcpy(temp,"rm ");
		strcat(temp,output_filename);
		system(temp);//to remove existing fits file
		bzero(temp,sizeof(temp));
			
 		if (fits_create_file(&fptr, output_filename, &status)) 
		{
        	 	printerror( status );           
			MPI_Finalize();
		}
		if ( fits_create_img(fptr,  bitpix, 0, 0, &status) )	
		{
        		printerror( status ); 	
			MPI_Finalize();
		}
		

		for(int i=0;i<npes;i++)
		{	getTimeEnergy("TimeEnergy",time,energy);
			MPI_Recv (x,totalGeneratedCount, MPI_INT, MPI_ANY_SOURCE,i+10, MPI_COMM_WORLD, &stat);
			MPI_Recv (y,totalGeneratedCount, MPI_INT, MPI_ANY_SOURCE,i+20, MPI_COMM_WORLD, &stat);
			MPI_Recv (detectorId,totalGeneratedCount, MPI_INT, MPI_ANY_SOURCE,i+30, MPI_COMM_WORLD, &stat);
			MPI_Recv (pixelNo,totalGeneratedCount, MPI_INT, MPI_ANY_SOURCE,i+40, MPI_COMM_WORLD, &stat);
			
			strcpy(extname,"Q");
			sprintf(temp,"%d",i);
			strcat(extname,temp);
			if ( fits_create_tbl( fptr, BINARY_TBL, nrows, tfields, ttype, tform,tunit, extname, &status) )
        	 		printerror( status );
			fits_write_col(fptr, TINT, 1, firstrow, firstelem, nrows, time,&status);
		        fits_write_col(fptr,TINT,2,firstrow, firstelem, nrows, energy,&status);				
			fits_write_col(fptr, TINT, 3, firstrow, firstelem, nrows, detectorId,&status);	
			fits_write_col(fptr, TINT, 4, firstrow, firstelem, nrows, pixelNo,&status);	
			fits_write_col(fptr, TINT, 5, firstrow, firstelem, nrows, x,&status);	
			fits_write_col(fptr, TINT, 6, firstrow, firstelem, nrows, y,&status);
		}
		if ( fits_close_file(fptr, &status) )                
        	printerror( status ); 
		timeSpent=MPI_Wtime()-timeSpent;
	}
	MPI_Finalize();
		
}

//Reading energy of event from input file
void getTimeEnergy(char * filename,int *time,int * energy)
{
	FILE *fp;
	int i=0;
	fp=fopen(filename,"r");
	if(fp==NULL)
	{
		printf("Error(%s:%d):Error while opening %s file\n",__FILE__,__LINE__,filename);
		exit(0);
	}
	for(i=0;i<TOTALACCEPTED;i++)
	{
		fscanf(fp,"%d",&time[i]);
		fscanf(fp,"%d",&energy[i]);
	}
	fclose(fp);
}

//Initilizing the cuda environment
void cudaInit(int gpuId)
{
	hipSetDevice(gpuId);
	float *initmalloc;
	hipMalloc(&initmalloc,sizeof(float));
}
//function to simulate event files
void executeKernel(float tx,float ty,int myrank,int gpuId,int *totalAccepted,int *totalRejected,int totalGeneratedCount)
{
	float height=481,*dphValues=NULL,*dphValues_device=NULL;
	int *maskPattern=NULL,*maskPattern_device=NULL,no_elements=64;
	int *acceptedCount=NULL,*rejectedCount=NULL,*acceptedCount_device=NULL,*rejectedCount_device=NULL;
	int i=0,j=0;
	char mask_fileName[100],temp[100];
	
	FILE *fp;
	hipError_t cuerr;
	
	int noOfBlock=noBlock,noOfThread=noThread;

	int totalThreads=noOfBlock*noOfThread;	
	int totalElements=totalThreads*no_elements*no_elements;
	int countPerThread=totalGeneratedCount/totalThreads;
	int remainingCount=totalGeneratedCount%totalThreads;
	int *x,*y,*pixelNo,*detectorId;
	int *x_device,*y_device,*pixelNo_device,*detectorId_device;
	/*
		set and initilise the cuda device
	*/
	
	cudaInit(gpuId);
	
	/*
		Allocation of the memory for the host and device
	*/

	x=(int*)malloc(sizeof(int*)*(totalGeneratedCount));
	y=(int*)malloc(sizeof(int*)*(totalGeneratedCount));
	pixelNo=(int*)malloc(sizeof(int*)*(totalGeneratedCount));
	detectorId=(int*)malloc(sizeof(int*)*(totalGeneratedCount));

	maskPattern=(int*)malloc(sizeof(int)*no_elements*no_elements);
	dphValues=(float*)malloc(sizeof(float)*totalElements);
	acceptedCount=(int*)malloc(sizeof(int)*totalThreads);
	rejectedCount=(int*)malloc(sizeof(int)*totalThreads);

	/*
		Here add loop for to execute it for each quadrant 
	*/
		strcpy(mask_fileName,"maskpattern/Q");
		sprintf(temp,"%d",myrank);
		strcat(mask_fileName,temp);
		strcat(mask_fileName,"mask.dat");
		
		fp=fopen(mask_fileName,"r");
		if(fp==NULL)
		{
			printf("Error(%s:%d):%s file not exist\n",__FILE__,__LINE__,mask_fileName);	
			exit(0);
		}
		for(i=0;i<no_elements;i++)
		{
			for(j=0;j<no_elements;j++)
			{
				fscanf(fp,"%d",&maskPattern[((63-i)*no_elements)+j]);
			}
		}
		fclose(fp);	
					
		/*
			Memory Allocation for device
		*/

		if((cuerr = hipGetLastError()) != hipSuccess)
		{
			printf("\nError:(Pre Malloc) \"%s\"\n", hipGetErrorString(cuerr));
		}

		hipMalloc(&x_device,sizeof(int*)*(totalGeneratedCount));
		hipMalloc(&y_device,sizeof(int*)*(totalGeneratedCount));
		hipMalloc(&pixelNo_device,sizeof(int*)*(totalGeneratedCount));
		hipMalloc(&detectorId_device,sizeof(int*)*(totalGeneratedCount));	
		hipMalloc(&maskPattern_device,sizeof(int)*no_elements*no_elements);
		hipMalloc(&dphValues_device,sizeof(float)*totalElements);
		hipMalloc(&acceptedCount_device,sizeof(int)*totalThreads);
		hipMalloc(&rejectedCount_device,sizeof(int)*totalThreads);
		if((cuerr = hipGetLastError()) != hipSuccess)
		{
			printf("\nError:(Pre CudaMemcpy) \"%s\"\n", hipGetErrorString(cuerr));
		}		
		//coping mask pattern
		hipMemcpy(maskPattern_device,maskPattern,sizeof(int)*no_elements*no_elements,hipMemcpyHostToDevice);
		if((cuerr = hipGetLastError()) != hipSuccess)
		{
			printf("\nError:(Post memcpy) \"%s\"\n", hipGetErrorString(cuerr));
		}
		//calling kernel
		kernel<<<noOfBlock,noOfThread>>>(tx,ty,height,maskPattern_device,dphValues_device,acceptedCount_device,rejectedCount_device,countPerThread,remainingCount,x_device,y_device,pixelNo_device,detectorId_device,myrank);

		//coping back the results
		hipMemcpy(dphValues,dphValues_device,sizeof(float)*totalElements,hipMemcpyDeviceToHost);
		hipMemcpy(acceptedCount,acceptedCount_device,sizeof(int)*totalThreads,hipMemcpyDeviceToHost);
		hipMemcpy(rejectedCount,rejectedCount_device,sizeof(int)*totalThreads,hipMemcpyDeviceToHost);

		hipMemcpy(x,x_device,sizeof(int)*totalGeneratedCount,hipMemcpyDeviceToHost);
		hipMemcpy(y,y_device,sizeof(int)*totalGeneratedCount,hipMemcpyDeviceToHost);
		hipMemcpy(pixelNo,pixelNo_device,sizeof(int)*totalGeneratedCount,hipMemcpyDeviceToHost);
		hipMemcpy(detectorId,detectorId_device,sizeof(int)*totalGeneratedCount,hipMemcpyDeviceToHost);

		if((cuerr = hipGetLastError()) != hipSuccess)
		{
			printf("\nError:(Post dph kernel) \"%s\"\n", hipGetErrorString(cuerr));
		}

		MPI_Send (x,totalGeneratedCount, MPI_INT, 0,myrank+10, MPI_COMM_WORLD );
		MPI_Send (y,totalGeneratedCount, MPI_INT, 0, myrank+20, MPI_COMM_WORLD );
		MPI_Send (pixelNo,totalGeneratedCount, MPI_INT, 0, myrank+30, MPI_COMM_WORLD );
		MPI_Send (detectorId,totalGeneratedCount, MPI_INT, 0, myrank+40, MPI_COMM_WORLD );
		
		//releasing the memory
		hipFree(&y_device);
		hipFree(&x_device);
		hipFree(&pixelNo_device);
		hipFree(&detectorId_device);
		hipFree(&maskPattern_device);
		hipFree(&dphValues_device);	
		hipFree(acceptedCount_device);	
		hipFree(rejectedCount_device);	
		free(&maskPattern);
		free(&acceptedCount);
		free(&rejectedCount);

}//end of main
__global__ void kernel(float tx,float ty,float height,int *maskPattern,float * dphValues,int *acceptedCount,int *rejectedCount,int countPerThread,int remainingCount,int*x,int*y,int*pixelNo,int*detectorId,int quadrantId)
{	int index=blockIdx.x*blockDim.x+threadIdx.x;
	//Generating event using ray tracing	
	generateEvent(tx,ty,height,maskPattern,&dphValues[index*64*64],&acceptedCount[index],&rejectedCount[index],countPerThread,&x[index*countPerThread],&y[index*countPerThread],&pixelNo[index*countPerThread],&detectorId[index*countPerThread],quadrantId);
}

//function to convert length to detector module and pixel number
__device__ void getDetectorIdPixelNo(int x,int y,int *moduleNo,int *pixelNo,int quadrantId)
{
	
	int xtemp=0,ytemp=0;
	int moduleRows=16,moduleCols=16;

	*moduleNo=(x/16)+((y/16)*4);
	if(*moduleNo<4)
		*moduleNo+=12;
	else if(*moduleNo>=4 && *moduleNo<8)
		*moduleNo+=4;
	else if(*moduleNo>=8 && *moduleNo<12)			
		*moduleNo-=4;
	else
		*moduleNo-=12;
		
	xtemp=x;
	ytemp=y;
	while(xtemp>=16)
		xtemp-=16;
	while(ytemp>=16)
		ytemp-=16;
	*pixelNo=(moduleRows-1-ytemp)*moduleCols+xtemp;
	if(quadrantId==1||quadrantId==2)
	{
		*moduleNo=15-*moduleNo;
	}
}

//function to perform ray tracing
__device__ void generateEvent(float tx,float ty,float height,int *maskPattern,float * dphValues,int *accepted,int *rejected,int countPerThread,int*x_id,int*y_id,int *pixelNo,int *detectorId ,int quadrantId )
{	
	float binSize=2.5;
	float xmin=0,ymin=0,thetaX=0.0,thetaY=0.0;
	float mask_lower_left_x=0,mask_lower_left_y=0,x=0,y=0,x_dect=0,y_dect=0;;
	int acceptedCount=0,rejectedCount=0,totalCount=0;
	thetaX=ty*(PI/180);
	thetaY=tx*(PI/180);
	float x_mask=0,y_mask=0;
	//inilizing cuda random 
	hiprandState localState;
	hiprand_init(0,clock(), 0, &localState);

	for(int i=0;i<64;i++)
	{
		for(int j=0;j<64;j++)
			dphValues[i*64+j]=0;
	}
	while(acceptedCount<countPerThread)
	{
		totalCount++;
		xmin=0;
		ymin=0;
		//generating random lengths
		x=hiprand_uniform (&localState);
		x*=159.5;
		
		y=hiprand_uniform (&localState);
		y*=159.5;
		//getting pixel on the mask plate		
		x_mask=(x)*10;
		x_mask=(int)x_mask;
		x_mask/=10;
				
		y_mask=y*10;
		y_mask=(int)y_mask;
		y_mask/=10;	
				
		while(1)
		{
			if(fmod(x_mask,binSize)<0.01)
				break;
			x_mask-=0.01;
		}
		while(1)
		{
			if(fmod(y_mask,binSize)<0.01)
				break;
			y_mask-=0.01;
		}
		
		x_mask*=10;
		x_mask=(int)x_mask;
		x_mask/=10;

		y_mask*=10;
		y_mask=(int)y_mask;
		y_mask/=10;		
		
	//checking mask is open  or close	
	if(maskPattern[(int)(((x_mask/binSize)*64)+(y_mask/binSize))]==1)
	{
	
		x_dect=x-(height*(tan(thetaX)));
		y_dect=y-(height*(tan(thetaY)));
		while(x>=((xmin+16)*binSize))
		{
			if(xmin==48.0f)
				break;
			xmin+=16;
		}
		while(y>=((ymin+16)*binSize))
		{
			if(ymin==48.0f)
				break;
			ymin+=16;
		}
		
		if(x_dect<(xmin*binSize)|| (((xmin+16)*binSize)-x_dect)<0.1||y_dect<(ymin*binSize)||(((ymin+16)*binSize)-y_dect)<0.1)
		{
			rejectedCount++;
		}	
		else
		{
			mask_lower_left_x=(x_dect)*10;
			mask_lower_left_x=(int)mask_lower_left_x;
			mask_lower_left_x/=10;
				
			mask_lower_left_y=y_dect*10;
			mask_lower_left_y=(int)mask_lower_left_y;
			mask_lower_left_y/=10;	
				
			while(1)
			{
				if(fmod(mask_lower_left_x,binSize)<0.01)
					break;
				mask_lower_left_x-=0.01;
			}
			while(1)
			{
				if(fmod(mask_lower_left_y,binSize)<0.01)
					break;
				mask_lower_left_y-=0.01;
			}
			mask_lower_left_x*=10;
			mask_lower_left_x=(int)mask_lower_left_x;
			mask_lower_left_x/=10;

			mask_lower_left_y*=10;
			mask_lower_left_y=(int)mask_lower_left_y;
			mask_lower_left_y/=10;
			/*Get Pixel No and detector Id*/
			x_id[acceptedCount]=(int)(mask_lower_left_y/2.5);
			y_id[acceptedCount]=(int)(mask_lower_left_x/2.5);
			
			getDetectorIdPixelNo(x_id[acceptedCount],y_id[acceptedCount],&detectorId[acceptedCount],&pixelNo[acceptedCount],quadrantId);
			acceptedCount++;
			
		}//end of else i.e., pixel is outside of the detector
	}
	else
	{
		rejectedCount++;
	}
	}//end of the while
	accepted[0]=acceptedCount;
	rejected[0]=rejectedCount;
	
}

void printerror( int status)
{
    if (status)
    {
       fits_report_error(stderr, status); 
       exit( status );    
    }
    return;
}

